#include "hip/hip_runtime.h"
//
// Created by zarko on 7/19/24.
//

#include "common/cuda_util.cuh"

namespace svr {

__global__ void
G_copy_submat(CRPTR(double) in, double *const out, const unsigned in_m, const unsigned out_m, const unsigned out_mn)
{
    CU_STRIDED_FOR_i(out_mn) out[i] = in[LDi(i, out_m, in_m)];
}

void copy_submat(CPTR(double) in, double *const out, const unsigned ldin, const unsigned in_start_m, const unsigned in_start_n, const unsigned in_end_m,
                 const unsigned in_end_n, const unsigned ldout, hipMemcpyKind kind, const hipStream_t stm)
{
#if 1
    cu_errchk(hipMemcpy2DAsync(out, ldout * sizeof(double), in + in_start_m + in_start_n * ldin, ldin * sizeof(double), (in_end_m - in_start_m) * sizeof(double),
                                in_end_n - in_start_n, kind, stm));
#else
    const auto out_m = in_end_m - in_start_m;
    const auto out_n = in_end_n - in_start_n;
    const unsigned out_mn = out_m * out_n;
    const auto start_offset = out_start_n * in_m + out_start_m;
    G_copy_submat<<<CU_BLOCKS_THREADS(out_mn), 0, strm>>>(in + start_offset, out, in_m, out_m, start_offset, out_mn);
#endif
}

NppStreamContext get_npp_context(const unsigned gpuid, const hipStream_t custream)
{
    NppStreamContext res;
#ifdef HETEROGENOUS_GPU_HW
    hipDeviceProp_t prop;
    cu_errchk(hipGetDeviceProperties(&prop, gpuid));
    res.nMultiProcessorCount = prop.multiProcessorCount;
    res.nMaxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
    res.nMaxThreadsPerBlock = prop.maxThreadsPerBlock;
    res.nSharedMemPerBlock = prop.sharedMemPerBlock;
    res.nCudaDevAttrComputeCapabilityMajor = prop.major;
    res.nCudaDevAttrComputeCapabilityMinor = prop.minor;
#else
    static auto prop = [gpuid]() {
        hipDeviceProp_t prop;
        cu_errchk(hipGetDeviceProperties(&prop, gpuid));
        return prop;
    } ();
    static const NppStreamContext C_npp_ctx {nullptr /* CUDA stream */, 0 /* GPU device ID */,
            prop.multiProcessorCount,
            prop.maxThreadsPerMultiProcessor,
            prop.maxThreadsPerBlock,
            prop.sharedMemPerBlock,
            prop.major,
            prop.minor};
    res = C_npp_ctx;
#endif
    res.hStream = custream;
    res.nCudaDeviceId = gpuid;
    cu_errchk(hipStreamGetFlags(custream, &res.nStreamFlags));
    return res;
}

}