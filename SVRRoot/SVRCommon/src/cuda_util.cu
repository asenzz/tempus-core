#include "hip/hip_runtime.h"
//
// Created by zarko on 7/19/24.
//

#include "common/cuda_util.cuh"

namespace svr {

__global__ void
G_copy_submat(CRPTR(double) in, double *const out, const unsigned in_m, const unsigned out_m, const unsigned out_mn)
{
    CU_STRIDED_FOR_i(out_mn) out[i] = in[LDi(i, out_m, in_m)];
}

void copy_submat(CPTR(double) in, double *const out, const unsigned ldin, const unsigned in_start_m, const unsigned in_start_n, const unsigned in_end_m,
                 const unsigned in_end_n, const unsigned ldout, hipMemcpyKind kind, const hipStream_t stm)
{
#if 1
    cu_errchk(hipMemcpy2DAsync(out, ldout * sizeof(double), in + in_start_m + in_start_n * ldin, ldin * sizeof(double), (in_end_m - in_start_m) * sizeof(double),
                                in_end_n - in_start_n, kind, stm));
#else
    const auto out_m = in_end_m - in_start_m;
    const auto out_n = in_end_n - in_start_n;
    const unsigned out_mn = out_m * out_n;
    const auto start_offset = out_start_n * in_m + out_start_m;
    G_copy_submat<<<CU_BLOCKS_THREADS(out_mn), 0, strm>>>(in + start_offset, out, in_m, out_m, start_offset, out_mn);
#endif
}

NppStreamContext get_npp_context(const unsigned gpuid, const hipStream_t custream)
{
    NppStreamContext ctx;
    ctx.hStream = custream;
    ctx.nCudaDeviceId = gpuid;

#ifdef HETEROGENOUS_GPU_HW
    hipDeviceProp_t prop;
    cu_errchk(hipGetDeviceProperties(&prop, gpuid));
#else
    static auto prop = [gpuid]() {
        hipDeviceProp_t prop;
        cu_errchk(hipGetDeviceProperties(&prop, gpuid));
        return prop;
    } ();
#endif
    ctx.nMultiProcessorCount = prop.multiProcessorCount;
    ctx.nMaxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
    ctx.nMaxThreadsPerBlock = prop.maxThreadsPerBlock;
    ctx.nSharedMemPerBlock = prop.sharedMemPerBlock;
    ctx.nCudaDevAttrComputeCapabilityMajor = prop.major;
    ctx.nCudaDevAttrComputeCapabilityMinor = prop.minor;
    cu_errchk(hipStreamGetFlags(custream, &ctx.nStreamFlags));
    return ctx;
}

}