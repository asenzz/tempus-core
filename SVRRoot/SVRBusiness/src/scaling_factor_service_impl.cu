//
// Created by zarko on 12/02/2025.
//

#include <thrust/reduce.h>
#include "ScalingFactorService.hpp"
#include "common/cuda_util.cuh"

namespace svr {
namespace business {

void ScalingFactorService::cu_scale_calc_I(RPTR(double) v, const size_t n, double &sf, double &dc, const hipStream_t custream, const hipblasHandle_t cublas_H)
{
    dc = thrust::reduce(thrust::cuda::par.on(custream), v, v + n) / n;
    thrust::transform(thrust::cuda::par.on(custream), v, v + n, v,[dc] __device__(const double x) { return x - dc; });
    cu_errchk(hipStreamSynchronize(custream));
    cb_errchk(hipblasDasum(cublas_H, n, v, 1, &sf));
    sf /= n;
    sf /= common::C_input_obseg_labels;
    thrust::transform(thrust::cuda::par.on(custream), v, v + n, v,[sf] __device__(const double x) { return x / sf; });
    cu_errchk(hipStreamSynchronize(custream));
}

}
}