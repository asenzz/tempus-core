#include "hip/hip_runtime.h"
#include <thrust/async/reduce.h>
#include <npp.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/async/reduce.h>

#include <cmath>
#include <thread>
#include <hipblas.h>
#include <magma_types.h>
#include <magma_v2.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include "common/compatibility.hpp"
#include "common/gpu_handler.tpp"
#include "common/cuda_util.cuh"
#include "cuqrsolve.cuh"
#include "common/constants.hpp"
#include "onlinesvr.hpp"
#include "thrust/detail/extrema.inl"
#include "cuda_path.hpp"

namespace svr {
namespace solvers {


void __global__
G_score_kernel(
        double *score,
        const double *__restrict kernel, const double *__restrict ref,
        const size_t M, const double norm_ker, const double norm_ref)
{
    const size_t thr_ix = threadIdx.x;
    const size_t g_thr_ix = thr_ix + blockIdx.x * common::C_cu_block_size;
    const size_t grid_size = common::C_cu_block_size * gridDim.x;

    double sum = 0;
    for (size_t i = g_thr_ix; i < M; i += grid_size) sum += kernel[i] * ref[i];

    __shared__ double _sh_sum[common::C_cu_block_size];
    _sh_sum[thr_ix] = sum;
    __syncthreads();

    for (size_t size = common::C_cu_block_size / 2; size > 0; size /= 2) {
        if (thr_ix >= size) continue;
        _sh_sum[thr_ix] += _sh_sum[thr_ix + size];
        __syncthreads();
    }

    if (thr_ix == 0) *score = _sh_sum[0] / (norm_ker * norm_ref);
}

template<const unsigned k_block_size> __global__ void
G_kernel_from_distances_symm(double *__restrict K, const double *__restrict dist, const size_t mm, const size_t m, const double divisor)
{
    const size_t g_thr_ix = threadIdx.x + blockIdx.x * k_block_size;
    if (g_thr_ix >= mm) return;
    const size_t col = g_thr_ix / m;
    K[g_thr_ix + col] = K[((g_thr_ix + col) % m) * m + col] = 1. - dist[g_thr_ix] / divisor;
}

__global__ void
G_kernel_from_distances_symm(double *__restrict K, const double *__restrict dist, const size_t mm, const size_t m, const double divisor)
{
    const size_t g_thr_ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (g_thr_ix >= mm) return;
    const size_t col = g_thr_ix / m;
    K[g_thr_ix + col] = K[((g_thr_ix + col) % m) * m + col] = 1. - dist[g_thr_ix] / divisor;
}

void kernel_from_distances_symm(double *K, const double *Z, const size_t m, const double gamma)
{
    LOG4_THROW("Buggy.");

    double *d_K, *d_Z;
    const auto mm = m * m;
    const auto mat_size = mm * sizeof(double);
    const common::gpu_context ctx;
    cu_errchk(hipSetDevice(ctx.phy_id()));
    cu_errchk(hipMalloc((void **) &d_K, mat_size));
    cu_errchk(hipMalloc((void **) &d_Z, mat_size));
    cu_errchk(hipMemcpy(d_Z, Z, mat_size, hipMemcpyHostToDevice));
    const auto half_mm = mm / 2;
    G_kernel_from_distances_symm<<<CUDA_THREADS_BLOCKS(half_mm)>>>(d_K, d_Z, mm / 2, m, gamma);
    cu_errchk(hipDeviceSynchronize());
    cu_errchk(hipMemcpy(K, d_K, mat_size, hipMemcpyDeviceToHost));
    cu_errchk(hipFree(d_K));
    cu_errchk(hipFree(d_Z));
}

__global__ void
G_kernel_from_distances(double *__restrict__ K, const double *__restrict__ Z, const size_t mn, const double divisor)
{
    CUDA_STRIDED_FOR_i(mn) K[i] = 1. - Z[i] / divisor;
}

__global__ void
G_kernel_from_distances_inplace(double *__restrict__ Kz, const size_t mn, const double divisor)
{
    CUDA_STRIDED_FOR_i(mn) Kz[i] = 1. - Kz[i] / divisor;
}

// K = 1 - Z / (2 * gamma * gamma)
void kernel_from_distances(double *K, const double *Z, const size_t m, const size_t n, const double gamma)
{
    double *d_K, *d_Z;
    const size_t mn = m * n;
    const size_t mat_size = mn * sizeof(double);
    const common::gpu_context ctx;
    cu_errchk(hipSetDevice(ctx.phy_id()));
    hipStream_t cu_stream;
    cu_errchk(hipStreamCreateWithFlags(&cu_stream, hipStreamNonBlocking));
    cu_errchk(hipMallocAsync((void **) &d_Z, mat_size, cu_stream));
    cu_errchk(hipMemcpyAsync(d_Z, Z, mat_size, hipMemcpyHostToDevice, cu_stream));
    cu_errchk(hipMallocAsync((void **) &d_K, mat_size, cu_stream));
    G_kernel_from_distances<<<CUDA_THREADS_BLOCKS(mn), 0, cu_stream>>>(d_K, d_Z, mn, DIST(gamma));
    cu_errchk(hipFreeAsync(d_Z, cu_stream));
    cu_errchk(hipMemcpyAsync(K, d_K, mat_size, hipMemcpyDeviceToHost, cu_stream));
    cu_errchk(hipFreeAsync(d_K, cu_stream));
    cu_errchk(hipStreamSynchronize(cu_stream));
    cu_errchk(hipStreamDestroy(cu_stream));
}

void kernel_from_distances_inplace(double *Kz, const size_t m, const size_t n, const double gamma)
{
    double *d_Kz;
    const size_t mn = m * n;
    const size_t mat_size = mn * sizeof(double);
    const common::gpu_context ctx;
    cu_errchk(hipSetDevice(ctx.phy_id()));
    hipStream_t cu_stream;
    cu_errchk(hipStreamCreateWithFlags(&cu_stream, hipStreamNonBlocking));
    cu_errchk(hipMallocAsync((void **) &d_Kz, mat_size, cu_stream));
    cu_errchk(hipMemcpyAsync(d_Kz, Kz, mat_size, hipMemcpyHostToDevice, cu_stream));
    G_kernel_from_distances_inplace<<<CUDA_THREADS_BLOCKS(mn), 0, cu_stream>>>(d_Kz, mn, DIST(gamma));
    cu_errchk(hipMemcpyAsync(Kz, d_Kz, mat_size, hipMemcpyDeviceToHost, cu_stream));
    cu_errchk(hipFreeAsync(d_Kz, cu_stream));
    cu_errchk(hipStreamSynchronize(cu_stream));
    cu_errchk(hipStreamDestroy(cu_stream));
}

#if 0
void kernel_from_distances_inplace(double *Kz, const size_t m, const size_t n, const double gamma)
{
    double *d_Kz;
    const size_t mn = m * n;
    const size_t mat_size = mn * sizeof(double);
    const common::gpu_context ctx;
    hipStream_t cu_stream;
    cu_errchk(hipSetDevice(ctx.phy_id()));
    cu_errchk(hipStreamCreateWithFlags(&cu_stream, hipStreamNonBlocking));
    cu_errchk(hipMallocAsync((void **) &d_Kz, mat_size, cu_stream));
    cu_errchk(hipMemcpyAsync(d_Kz, Kz, mat_size, hipMemcpyHostToDevice, cu_stream));
    G_kernel_from_distances_inplace<<<CUDA_THREADS_BLOCKS(mn), 0, cu_stream>>>(d_Kz, mn, 2. * gamma * gamma);
    cu_errchk(hipMemcpyAsync(Kz, d_Kz, mat_size, hipMemcpyDeviceToHost, cu_stream));
    cu_errchk(hipFreeAsync(d_Kz, cu_stream));
    cu_errchk(hipStreamSynchronize(cu_stream));
    cu_errchk(hipStreamDestroy(cu_stream));
}
#endif

double
score_kernel(
        const double *ref_kernel /* colmaj order */,
        const double norm_ref,
        const double *Z /* colmaj order */,
        const size_t m,
        const double gamma)
{
#if 0
    double *d_K, *d_Z, *d_ref;
    const size_t mm = m * m;
    const size_t mat_size = mm * sizeof(double);

    const common::gpu_context ctx;
    cu_errchk(hipSetDevice(ctx.phy_id()));
    cu_errchk(hipMalloc(&d_K, mat_size));
    cu_errchk(hipMalloc(&d_Z, mat_size));
    cu_errchk(hipMemcpy(d_Z, Z, mat_size, hipMemcpyHostToDevice));
    G_kernel_from_distances_symm<CUDA_THREADS(mm)><<<CUDA_THREADS_BLOCKS(mm)>>>(d_K, d_Z, mm, m, 2. * gamma * gamma);
    cu_errchk(hipFree(d_Z));
    cu_errchk(hipDeviceSynchronize());
    hipblasHandle_t cublasH;
    cublas_safe_call(hipblasCreate(&cublasH));
    double norm_K;
    cublas_safe_call(hipblasDnrm2(cublasH, mm, d_K, m, &norm_K));
    cu_errchk(hipDeviceSynchronize());
    double *d_score;
    cu_errchk(hipMalloc(&d_score, sizeof(double)));
    cu_errchk(hipMalloc(&d_ref, mat_size));
    cu_errchk(hipMemcpy(d_ref, ref_kernel, mat_size, hipMemcpyHostToDevice));
    G_score_kernel<<<CUDA_THREADS_BLOCKS(m)>>>(d_score, d_K, d_ref, mm, norm_K, norm_ref);
    double score;
    cu_errchk(hipMemcpy(&score, d_score, sizeof(double), hipMemcpyDeviceToHost));
    cu_errchk(hipFree(d_score));
    cu_errchk(hipFree(d_ref));
    cu_errchk(hipFree(d_K));
    cublas_safe_call(hipblasDestroy(cublasH));
    return 2. - score;
#endif
    return 0;
}


void __global__
gpu_copy_upper_submatrix(
        const double *__restrict d_in,
        double *__restrict d_ou,
        const size_t M, const size_t N, const size_t subM)
{
    const size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= subM || j >= N) return;
    d_ou[j * subM + i] = d_in[j * M + i];
}


std::tuple<hipsolverHandle_t, double *, double *, double *, int *, int *>
init_cusolver(const size_t gpu_id, const size_t m, const size_t n)
{
    cu_errchk(hipSetDevice(gpu_id));

    hipsolverHandle_t cusolverH;
    hipblasHandle_t cublasH;
    int lwork;
    double *d_Ainput, *d_B, *d_work;
    int *d_Ipiv, *d_devInfo;

    cs_errchk(hipsolverDnCreate(&cusolverH));
    cb_errchk(hipblasCreate(&cublasH));
    cu_errchk(hipMalloc((void **) &d_Ainput, m * m * sizeof(double)));
    cu_errchk(hipMalloc((void **) &d_B, m * n * sizeof(double)));
    cs_errchk(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_Ainput, m /* lda */, &lwork));
    cu_errchk(hipMalloc((void **) &d_work, sizeof(double) * lwork));
    cu_errchk(hipMalloc((void **) &d_Ipiv, m * sizeof(int)));
    cu_errchk(hipMalloc((void **) &d_devInfo, sizeof(int)));

    return {cusolverH, d_Ainput, d_B, d_work, d_Ipiv, d_devInfo};
}


void uninit_cusolver(const size_t gpu_id, const hipsolverHandle_t cusolverH, double *d_Ainput, double *d_B, double *d_work, int *d_Ipiv, int *d_devInfo)
{
    cu_errchk(hipSetDevice(gpu_id));

    if (d_Ainput) cu_errchk(hipFree(d_Ainput));
    if (d_B) cu_errchk(hipFree(d_B));
    if (d_work) cu_errchk(hipFree(d_work));
    if (d_Ipiv) cu_errchk(hipFree(d_Ipiv));
    if (d_devInfo) cu_errchk(hipFree(d_devInfo));

    if (cusolverH) cs_errchk(hipsolverDnDestroy(cusolverH));
}

void dyn_gpu_solve(const hipsolverHandle_t cusolver_H, const size_t m, const size_t n, const double *d_a, double *d_b, double *d_work, int *d_piv, int *d_info)
{
    cs_errchk(hipsolverDnDgetrf(cusolver_H, m, m, (double *) d_a, m, d_work, d_piv, d_info));
    cs_errchk(hipsolverDnDgetrs(cusolver_H, HIPBLAS_OP_N, m, n, d_a, m, d_piv, d_b, m, d_info));
}

void h_dyn_gpu_solve(
        const size_t gpu_id, const size_t m, const size_t n, const double *h_K, const double *h_L, double *h_weights, hipsolverHandle_t cusolver_H,
        double *d_a, double *d_b, double *d_work, int *d_piv, int *d_info)
{
    cu_errchk(hipSetDevice(gpu_id));
    cu_errchk(hipMemcpy(d_a, h_K, sizeof(double) * m * m, hipMemcpyHostToDevice));
    cu_errchk(hipMemcpy(d_b, h_L, sizeof(double) * m * n, hipMemcpyHostToDevice));
    dyn_gpu_solve(cusolver_H, m, n, d_a, d_b, d_work, d_piv, d_info);
    cu_errchk(hipMemcpy(h_weights, d_b, sizeof(double) * m * n, hipMemcpyDeviceToHost));
}


std::tuple<magma_queue_t, magmaDouble_ptr, magmaDouble_ptr, magmaDouble_ptr, magmaDouble_ptr, magmaFloat_ptr, magmaInt_ptr>
init_magma_solver(const size_t m, const size_t b_n, const bool psd, const size_t gpu_id)
{
    cu_errchk(hipSetDevice(gpu_id));
    magma_queue_t magma_queue;
    magma_queue_create(gpu_id, &magma_queue);
    if (!magma_queue) LOG4_THROW("Failed creating MAGMA queue.");

    magmaDouble_ptr d_a, d_b, d_x, d_wd;
    magmaFloat_ptr d_ws;
    auto piv = (magmaInt_ptr) malloc(m * sizeof(magma_int_t)); // host mem.
    ma_errchk(magma_dmalloc(&d_a, m * m));
    ma_errchk(magma_dmalloc(&d_b, m * b_n));
    if (psd) {
        ma_errchk(magma_dmalloc(&d_x, m * b_n));
        ma_errchk(magma_dmalloc(&d_wd, m * (m + b_n) + m));
        ma_errchk(magma_smalloc(&d_ws, m * (m + b_n) + m));
    } else {
        d_x = nullptr;
        d_wd = nullptr;
        d_ws = nullptr;
    }
    return {magma_queue, d_a, d_b, d_x, d_wd, d_ws, piv};
}

std::tuple<std::vector<magmaDouble_ptr>, std::vector<magmaDouble_ptr>>
init_magma_batch_solver(const size_t batch_size, const size_t m, const size_t n)
{
    std::vector<magmaDouble_ptr> d_a(batch_size, nullptr), d_b(batch_size, nullptr);
#pragma omp unroll
    for (size_t i = 0; i < batch_size; ++i) {
        ma_errchk(magma_dmalloc(&d_a[i], m * m));
        ma_errchk(magma_dmalloc(&d_b[i], m * n));
    }
    return {d_a, d_b};
}


void uninit_magma_solver(
        const magma_queue_t &magma_queue,
        const magmaDouble_ptr d_a, const magmaDouble_ptr d_b, const magmaDouble_ptr d_x, const magmaDouble_ptr d_wd, const magmaFloat_ptr d_ws, const magmaInt_ptr piv,
        const size_t gpu_id)
{
    cu_errchk(hipSetDevice(gpu_id));
    if (d_a) ma_errchk(magma_free(d_a));
    if (d_b) ma_errchk(magma_free(d_b));
    if (d_x) ma_errchk(magma_free(d_x));
    if (d_wd) ma_errchk(magma_free(d_wd));
    if (d_ws) ma_errchk(magma_free(d_ws));
    if (piv) free(piv);

    if (magma_queue) magma_queue_destroy(magma_queue);
}


void uninit_magma_batch_solver(std::vector<magmaDouble_ptr> &d_a, std::vector<magmaDouble_ptr> &d_b)
{
#pragma omp unroll
    for (size_t i = 0; i < d_a.size(); ++i) {
        if (d_a[i]) ma_errchk(magma_free(d_a[i]));
        if (d_b[i]) ma_errchk(magma_free(d_b[i]));
    }
}


void iter_magma_solve(
        const int m, const int b_n, const double *a, const double *b, double *output, const magma_queue_t magma_queue,
        const magmaDouble_ptr d_a, const magmaDouble_ptr d_b, const magmaDouble_ptr d_x, const magmaDouble_ptr d_workd,
        const magmaFloat_ptr d_works, const bool psd, const size_t gpu_id)
{
    cu_errchk(hipSetDevice(gpu_id));
    magma_int_t err, iter, info;

    magma_dsetmatrix(m, m, a, m, d_a, m, magma_queue); // copy a -> d_a
    magma_dsetmatrix(m, b_n, b, m, d_b, m, magma_queue); // copy b -> d_b

    if (!psd) goto __solve_dgesv;

    if ((err = magma_dshposv_gpu_expert(magma_uplo_t::MagmaLower, m, b_n, d_a, m, d_b, m, d_x, m, d_workd, d_works, &iter, magma_mode_t::MagmaHybrid, 1, 0, 0, 0,
                                        &info)) < MAGMA_SUCCESS || info != 0) {
        LOG4_WARN("Call to magma_dshposv_gpu_expert failed with error " << err << ", info " << info << ". Trying magma_dgesv_rbt.");
        if (iter < 0) {
            switch (iter) {
                case -1:
                    LOG4_DEBUG(
                            "Iterative magma_dshposv_gpu_expert returned -1 : the routine fell back to full precision for implementation - or machine-specific reasons");
                    break;
                case -2:
                    LOG4_DEBUG("Iterative magma_dshposv_gpu_expert returned -2 : narrowing the precision induced an overflow, the routine fell back to full precision");
                    break;
                case -3:
                    LOG4_DEBUG("Iterative magma_dshposv_gpu_expert returned -3 : failure of SPOTRF");
                    break;
                case -31:
                    LOG4_DEBUG("Iterative magma_dshposv_gpu_expert returned -31: stop the iterative refinement after the 30th iteration");
                    break;
                default:
                    LOG4_ERROR("Iterative refinement magma_dshposv_gpu_expert has failed, double precision factorization has been performed");
            }
        }
    } else {
        LOG4_TRACE("Call to magma_dshposv_gpu_expert triunfo.");
        magma_dgetmatrix(m, b_n, d_x, m, output, m, magma_queue);
        return;
    }

    __solve_dgesv:
    ma_errchk(magma_dgesv_rbt_q(magma_bool_t::MagmaTrue, m, b_n, d_a, m, d_b, m, &info, datamodel::C_rbt_iter, datamodel::C_rbt_threshold, magma_queue));
    if (psd) LOG4_DEBUG("Call to magma_dgesv_rbt triunfo.");
    magma_dgetmatrix(m, b_n, d_b, m, output, m, magma_queue); // copy solution d_b -> output
}

void iter_magma_solve(
        const int m, const int n, const double *a, const double *b, double *output, const magma_queue_t &magma_queue,
        const magmaDouble_ptr d_a, const magmaDouble_ptr d_b)
{
    magma_int_t info;
    magma_dsetmatrix(m, m, a, m, d_a, m, magma_queue);
    magma_dsetmatrix(m, n, b, m, d_b, m, magma_queue);
    cu_errchk(hipDeviceSynchronize());
    ma_errchk(magma_dgesv_rbt_q(magma_bool_t::MagmaTrue, m, n, d_a, m, d_b, m, &info, datamodel::C_rbt_iter, datamodel::C_rbt_threshold, magma_queue));
    cu_errchk(hipDeviceSynchronize());
    magma_dgetmatrix(m, n, d_b, m, output, m, magma_queue);
    cu_errchk(hipDeviceSynchronize());
}

void iter_magma_batch_solve(
        const int m, const int n, const std::deque<arma::mat> &a, const std::deque<arma::mat> &b, std::deque<arma::mat> &output,
        const magma_queue_t magma_queue, std::vector<magmaDouble_ptr> &d_a, std::vector<magmaDouble_ptr> &d_b, const size_t gpu_id)
{
    const auto batch_size = a.size();
    LOG4_DEBUG("m " << magma_int_t(m) << ", n " << magma_int_t(n) << ", batch size " << batch_size);
// #pragma omp parallel for schedule(static, 1) num_threads(adj_threads(batch_size))
    cu_errchk(hipSetDevice(gpu_id));
#pragma omp unroll
    for (size_t i = 0; i < batch_size; ++i) {
        magma_dsetmatrix(m, m, a[i].mem, m, d_a[i], m, magma_queue);
        magma_dsetmatrix(m, n, b[i].mem, m, d_b[i], m, magma_queue);
    }
    std::vector<magma_int_t> info(batch_size);
    cu_errchk(hipSetDevice(gpu_id));
    auto da_data = &d_a[0];
    auto db_data = &d_b[0];
    ma_errchk(magma_dgesv_rbt_batched(magma_int_t(m), magma_int_t(n), da_data, m, db_data, m, info.data(), magma_int_t(batch_size), magma_queue));

#pragma omp parallel for schedule(static, 1) num_threads(adj_threads(batch_size))
    for (size_t i = 0; i < batch_size; ++i) {
        cu_errchk(hipSetDevice(gpu_id));
        magma_dgetmatrix(m, n, d_b[i], m, output[i].memptr(), m, magma_queue); // copy solution d_b -> output
    }
}

// Doesn't work with NVidia CuSolver 12.1, leaks memory
void dyn_magma_solve(const int m, const int b_n, const double *a, const double *b, double *output, magma_queue_t magma_queue,
                     const magmaInt_ptr piv, const magmaDouble_ptr d_a, const magmaDouble_ptr d_b, const size_t gpu_id)
{
    cu_errchk(hipSetDevice(gpu_id));
    magma_int_t info, err;
    magma_dsetmatrix(m, m, a, m, d_a, m, magma_queue);

    // find the inverse matrix: d_a*X=I using the LU factorization
    // with partial pivoting and row interchanges computed by
    // magma_dgetrf_gpu; row i is interchanged with row piv(i);
    // d_a -mxm matrix; d_a is overwritten by the inverse
    const magma_int_t nb = magma_get_dgetrf_native_nb(m, b_n);
    if ((err = magma_dgetrf_gpu_expert(m, m, d_a, m, piv, &info, nb, magma_mode_t::MagmaNative)) < MAGMA_SUCCESS)
        LOG4_THROW("Failed calling magma_dgetrf_gpu with error code " << err << ", info " << info);

    magma_dsetmatrix(m, b_n, b, m, d_b, m, magma_queue);
    if ((err = magma_dgetrs_gpu(magma_trans_t::MagmaNoTrans, m, b_n, d_a, m /* lda */, piv, d_b, m /* ldb */, &info)) < MAGMA_SUCCESS)
        LOG4_THROW("Failed calling magma_dgetrs_gpu with error code " << err << ", info " << info);

    magma_dgetmatrix(m, b_n, d_b, m, output, m, magma_queue); // copy solution d_b -> output
}


void
qrsolve_over(const size_t Nrows, const size_t Ncols, const size_t Nrhs, double *d_Ainput, double *d_b, double *d_output)
{
    // define handles
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    const size_t M = Nrows;
    const size_t N = Ncols;
    const size_t K = Nrhs;

    cs_errchk(hipsolverDnCreate(&cusolverH));
    cb_errchk(hipblasCreate(&cublasH));

    int *d_devInfo;
    double *d_tau;
    double *d_work;
    double *d_work2;
    cu_errchk(hipMalloc(&d_tau, sizeof(double) * M));
    cu_errchk(hipMalloc(&d_devInfo, sizeof(int)));
    int bufSize, bufSize2;

    // in-place A = QR
    cs_errchk(
            hipsolverDnDgeqrf_bufferSize(
                    cusolverH,
                    M,
                    N,
                    d_Ainput,
                    M,
                    &bufSize
            )
    );
    cu_errchk(hipMalloc(&d_work, sizeof(double) * bufSize));
    cs_errchk(
            hipsolverDnDgeqrf(
                    cusolverH,
                    M,
                    N,
                    d_Ainput,
                    M,
                    d_tau,
                    d_work,
                    bufSize,
                    d_devInfo
            )
    );

    // Q^T*b
    cs_errchk(
            hipsolverDnDormqr_bufferSize(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    N,
                    d_Ainput,
                    M,
                    d_tau,
                    d_b,
                    M,
                    &bufSize2
            )
    );

    cu_errchk(hipMalloc(&d_work2, sizeof(double) * bufSize2));
    cs_errchk(
            hipsolverDnDormqr(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    std::min<size_t>(M, N),
                    d_Ainput,
                    M,
                    d_tau,
                    d_b,
                    M,
                    d_work2,
                    bufSize2,
                    d_devInfo
            )
    );

    // need to explicitly copy submatrix for the triangular solve
    double *d_R;
    cu_errchk(hipMalloc(&d_R, sizeof(double) * N * N));
    dim3 thd_size(32, 32);
    dim3 blk_size((N + thd_size.x - 1) / thd_size.x, (N + thd_size.y - 1) / thd_size.y);
    gpu_copy_upper_submatrix<<<blk_size, thd_size>>>(d_Ainput, d_R, M, N, N);
    blk_size = dim3((N + thd_size.x - 1) / thd_size.x, (K + thd_size.y - 1) / thd_size.y);
    gpu_copy_upper_submatrix<<<blk_size, thd_size>>>(d_b, d_output, M, K, N);

    // solve x = R \ (Q^T*B)
    const double one = 1;
    cb_errchk(
            hipblasDtrsm(
                    cublasH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_FILL_MODE_UPPER,
                    HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT,
                    N,
                    K,
                    &one,
                    d_R,
                    N,
                    d_output,
                    N
            )
    );

    cu_errchk(hipFree(d_R));
    cu_errchk(hipFree(d_work));
    cu_errchk(hipFree(d_work2));
    cu_errchk(hipFree(d_tau));
    cb_errchk(hipblasDestroy(cublasH));
    cs_errchk(hipsolverDnDestroy(cusolverH));
}


void
call_gpu_overdetermined(
        const size_t Nrows, const size_t Ncols, const size_t Nrhs, const double *cpu_matrix,
        const double *cpu_rhs, double *cpu_output)
{
    const svr::common::gpu_context gtx;
    hipSetDevice(gtx.phy_id());

    thrust::device_vector<double> gpu_matrix(Nrows * Ncols);
    thrust::device_vector<double> gpu_rhs(Nrows * Nrhs);
    thrust::device_vector<double> gpu_output(Ncols * Nrhs);
    cu_errchk(hipMemcpy(thrust::raw_pointer_cast(gpu_matrix.data()), cpu_matrix, sizeof(double) * Nrows * Ncols,
                         hipMemcpyHostToDevice));
    cu_errchk(hipMemcpy(thrust::raw_pointer_cast(gpu_rhs.data()), cpu_rhs, sizeof(double) * Nrows * Nrhs,
                         hipMemcpyHostToDevice));
    qrsolve_over(Nrows, Ncols, Nrhs, thrust::raw_pointer_cast(gpu_matrix.data()),
                 thrust::raw_pointer_cast(gpu_rhs.data()), thrust::raw_pointer_cast(gpu_output.data()));
    cu_errchk(hipMemcpy(cpu_output, thrust::raw_pointer_cast(gpu_output.data()), sizeof(double) * Ncols * Nrhs,
                         hipMemcpyDeviceToHost));
}


// Adds err + addtive to K and solved labels
__global__ void G_irwls_op2(
        const double *__restrict__ err,
        const double *__restrict__ K,
        const double *__restrict__ labels,
        double *__restrict__ out_K,
        double *__restrict__ solved,
        const double additive,
        const unsigned m,
        const unsigned mn,
        const unsigned mm)
{
    double sum_err_i;
    CUDA_STRIDED_FOR_i(mm) {
        if (i < mn) solved[i] = (err[i] + additive) * labels[i];
        out_K[i] = K[i];
        sum_err_i = 0;
#ifdef PRODUCTION_BUILD
#pragma unroll
#endif
        for (unsigned j = i % m; j < mn; j += m) sum_err_i += err[j] + additive;
        out_K[i] *= sum_err_i;
    }
}

/* LDA version
__global__ void G_irwls_op2(
        const double *__restrict__ err,
        const double *__restrict__ K,
        const unsigned ldK,
        const double *__restrict__ labels,
        double *__restrict__ out_K,
        double *__restrict__ solved,
        const double additive,
        const unsigned m,
        const unsigned mn,
        const unsigned mm)
{
    double sum_err_i;
    CUDA_STRIDED_FOR_i(mm) {
        const unsigned row = i % m;
        const unsigned in_i = (i / m) * ldK + row;
        if (i < mn) solved[i] = (err[i] + additive) * labels[in_i];
        sum_err_i = 0;
#ifdef PRODUCTION_BUILD
#pragma unroll
#endif
        for (unsigned j = row; j < mn; j += m) sum_err_i += err[j] + additive;
        out_K[i] = K[in_i] * sum_err_i;
    }
}
*/

void solve_hybrid(
        const double *const j_K_epsco, const unsigned n, const unsigned train_len, double *const j_solved, const unsigned magma_iters, const double magma_threshold,
        const magma_queue_t ma_queue, const unsigned irwls_iters, const double *const j_train_labels, const size_t train_n_size,
        double *const j_train_error, const hipStream_t custream, const hipblasHandle_t cublas_H, const double *const j_K_tune, const double labels_factor,
        const size_t train_len_n, double &best_solve_score, unsigned &best_iter, double *const d_best_weights, const size_t K_train_len, double *const j_left,
        magma_int_t &info, const double iters_mul)
{
    constexpr double one = 1, oneneg = -1;
    cu_errchk(hipMemcpyAsync(j_solved, j_train_labels, train_n_size, hipMemcpyDeviceToDevice, custream));
    ma_errchk(magma_dgesv_rbt_q(MagmaTrue, train_len, n, (double *)j_K_epsco, train_len, j_solved, train_len, &info, magma_iters, magma_threshold, ma_queue));
#pragma unroll common::C_default_online_iter_limit
    for (size_t i = 1; i < irwls_iters + 1; ++i) {
        cu_errchk(hipMemcpyAsync(j_train_error, j_train_labels, train_n_size, hipMemcpyDeviceToDevice, custream));
        cb_errchk(hipblasDgemm(cublas_H, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              train_len, n, train_len, &one, (double *)j_K_tune, train_len, j_solved, train_len, &oneneg, j_train_error, train_len));
        const auto solve_score = labels_factor * solvers::irwls_op1(j_train_error, train_len_n, custream);
        if (!std::isnormal(solve_score))
            LOG4_THROW("Score not normal " << solve_score << ", iteration " << i << ", train len " << train_len);
        else if (solve_score < best_solve_score) {
            /* LOG4_TRACE("Try " << j << ", IRWLS iteration " << i << ", kernel dimensions " << train_len << "x" << train_len << ", former best score " <<
                            best_solve_score << ", new best score " << solve_score << ", improvement " << 100. * (1. - solve_score / best_solve_score) << " pct."); */
            best_solve_score = solve_score;
            best_iter = i;
            cu_errchk(hipMemcpyAsync(d_best_weights, j_solved, train_n_size, hipMemcpyDeviceToDevice, custream));
            cu_errchk(hipStreamSynchronize(custream)); // TODO Remove
        }
        if (i == irwls_iters) break;
        G_irwls_op2<<<CUDA_THREADS_BLOCKS(K_train_len), 0, custream>>>(
                j_train_error, j_K_epsco, j_train_labels, j_left, j_solved, common::C_itersolve_delta / (double(i) * iters_mul), train_len, train_len_n, K_train_len);
        /*
        G_irwls_op2<<<CUDA_THREADS_BLOCKS(K_train_len), 0, custream>>>(
        j_train_error, d_K_epsco + train_start, m, d_tune_labels + train_start, j_left, j_solved, common::C_itersolve_delta / (double(i) * iters_mul),
        train_len, train_len_n, K_train_len);*/
        ma_errchk(magma_dgesv_rbt_q(MagmaTrue, train_len, n, j_left, train_len, j_solved, train_len, &info, magma_iters, magma_threshold, ma_queue));
    }
}


__global__ void G_abs(double *__restrict__ inout, const size_t N)
{
    CUDA_STRIDED_FOR_i(N)inout[i] = _ABS(inout[i]);
}


template<const unsigned block_size> __device__ inline void
warp_reduce_sumabs(volatile double *sumdata, const unsigned ix, const unsigned n)
{
#define _DO_WARP_REDUCE_SUMABS(N)               \
    if (block_size >= (N)) {                    \
        const unsigned ix_N_2 = ix + (N) / 2;   \
        if (ix_N_2 < n)                         \
            sumdata[ix] += sumdata[ix_N_2];     \
    }

    _DO_WARP_REDUCE_SUMABS(64);
    _DO_WARP_REDUCE_SUMABS(32);
    _DO_WARP_REDUCE_SUMABS(16);
    _DO_WARP_REDUCE_SUMABS(8);
    _DO_WARP_REDUCE_SUMABS(4);
    _DO_WARP_REDUCE_SUMABS(2);
}

template<const unsigned block_size> __global__ void
G_sumabs(const double *__restrict__ d_input, double *__restrict__ d_result_sum, const size_t n)
{
    extern __shared__ double sumdata[];
    auto i = blockIdx.x * block_size + tid;
    if (i < n) {
        sumdata[tid] = fabs(d_input[i]);
        const auto stride1 = blockDim.x * gridDim.x;
#pragma unroll
        for (i += stride1; i < n; i += stride1) sumdata[tid] += fabs(d_input[i]);
    } else
        sumdata[tid] = 0;

    __syncthreads();
    const auto sh_limit = _MIN(n, block_size);
#define stride_reduce_sum(block_low_)                        \
        if (block_size >= block_low_) {                      \
            constexpr unsigned stride2 = block_low_ / 2;     \
            const auto tid_stride2 = tid + stride2;          \
            if (tid < stride2 && tid_stride2 < sh_limit)     \
                sumdata[tid] += sumdata[tid_stride2];        \
            __syncthreads();                                 \
        }

    stride_reduce_sum(1024);
    stride_reduce_sum(512);
    stride_reduce_sum(256);
    stride_reduce_sum(128);
    if (tid >= 32) return;
    warp_reduce_sumabs<block_size>(sumdata, tid, sh_limit);
    if (tid) return;
    d_result_sum[blockIdx.x] = sumdata[0];
}

double sumabs(const double *d_in, const size_t n, const hipStream_t &stm)
{
    double sum;
    const auto clamped_n = clamp_n(n);
    const auto grid_len = CUDA_BLOCKS(clamped_n);
    const auto threads = CUDA_THREADS(clamped_n);
    const auto grid_size = grid_len * sizeof(double);
    double *d_sum;
    cu_errchk(hipMallocAsync((void **) &d_sum, grid_size, stm));
    G_sumabs<common::C_cu_block_size><<<grid_len, threads, threads * sizeof(double), stm>>>(d_in, d_sum, n);
    if (grid_len > 1)
        sum = thrust::async::reduce(thrust::cuda::par.on(stm), d_sum, d_sum + grid_len, double(0), thrust::plus<double>()).get();
    else
        cu_errchk(hipMemcpyAsync(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipFreeAsync(d_sum, stm));
    cu_errchk(hipStreamSynchronize(stm));
    return sum;
}

template<const unsigned block_size> __global__ void
G_irwls_op1(double *__restrict__ d_input, double *__restrict__ d_result_sum, const size_t n)
{
    __shared__ double sumdata[block_size];
    auto i = blockIdx.x * block_size + tid;
    if (i < n) {
        d_input[i] = fabs(d_input[i]);
        sumdata[tid] = d_input[i];
        const auto stride1 = blockDim.x * gridDim.x;
#pragma unroll
        for (i += stride1; i < n; i += stride1) {
            d_input[i] = fabs(d_input[i]);
            sumdata[tid] += d_input[i];
        }
    } else
        sumdata[tid] = 0;

    __syncthreads();
    const auto sh_limit = _MIN(n, block_size);
#define stride_reduce_sum(block_low_)                        \
        if (block_size >= block_low_) {                      \
            constexpr unsigned stride2 = block_low_ / 2;     \
            const auto tid_stride2 = tid + stride2;          \
            if (tid < stride2 && tid_stride2 < sh_limit)     \
                sumdata[tid] += sumdata[tid_stride2];        \
            __syncthreads();                                 \
        }

    stride_reduce_sum(1024);
    stride_reduce_sum(512);
    stride_reduce_sum(256);
    stride_reduce_sum(128);
    if (tid >= 32) return;
    warp_reduce_sumabs<block_size>(sumdata, tid, sh_limit);
    if (tid) return;
    d_result_sum[blockIdx.x] = sumdata[0];
}

// Returns meanabs of input, input = abs(input)
double irwls_op1(double *d_in, const size_t n, const hipStream_t &stm)
{
    double sum;
    const auto clamped_n = clamp_n(n);
    const auto grid_len = CUDA_BLOCKS(clamped_n);
    const auto threads = CUDA_THREADS(clamped_n);
    const auto grid_size = grid_len * sizeof(double);
    double *d_sum;
    cu_errchk(hipMallocAsync((void **) &d_sum, grid_size, stm));
    G_irwls_op1 < common::C_cu_block_size ><<<grid_len, threads, 0, stm>>>(d_in, d_sum, n);
    if (grid_len > 1)
        sum = thrust::async::reduce(thrust::cuda::par.on(stm), d_sum, d_sum + grid_len, double(0), thrust::plus<double>()).get();
    else
        cu_errchk(hipMemcpyAsync(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipFreeAsync(d_sum, stm));
    cu_errchk(hipStreamSynchronize(stm));
    return sum / double(n); // Return mean
}

double meanabs(const double *d_in, const size_t n, const hipStream_t &stm)
{
    return sumabs(d_in, n, stm) / double(n);
}

#define _SMM_OP(X1, X2, X3, Y1, Y2, Y3) {       \
        (X1) += (Y1);                           \
        _MINAS((X2), (Y2));                     \
        _MAXAS((X3), (Y3));                     \
    }

template<const unsigned block_size> __device__ inline void
warp_reduce_suminmax(volatile double *sumdata, volatile double *mindata, volatile double *maxdata, const unsigned ix, const unsigned n)
{
#define _DO_WARP_REDUCE(N)                      \
    if (block_size >= (N)) {                    \
        const unsigned ix_N_2 = ix + (N) / 2;   \
        if (ix_N_2 < n)                         \
            _SMM_OP(sumdata[ix], mindata[ix], maxdata[ix], sumdata[ix_N_2], mindata[ix_N_2], maxdata[ix_N_2]); \
    }

    _DO_WARP_REDUCE(64);
    _DO_WARP_REDUCE(32);
    _DO_WARP_REDUCE(16);
    _DO_WARP_REDUCE(8);
    _DO_WARP_REDUCE(4);
    _DO_WARP_REDUCE(2);
}

template<const unsigned block_size> __global__ void
G_suminmax(const double *__restrict__ d_input, double *__restrict__ d_result_sum, double *__restrict__ d_result_min, double *__restrict__ d_result_max, const size_t n)
{
    __shared__ double sumdata[block_size], mindata[block_size], maxdata[block_size];
    auto i = blockIdx.x * block_size + tid;
    if (i < n) {
        sumdata[tid] = mindata[tid] = maxdata[tid] = d_input[i];
        const auto stride1 = blockDim.x * gridDim.x;
#pragma unroll
        for (i += stride1; i < n; i += stride1) _SMM_OP(sumdata[tid], mindata[tid], maxdata[tid], d_input[i], d_input[i], d_input[i]);
    } else {
        sumdata[tid] = 0;
        mindata[tid] = std::numeric_limits<double>::max();
        maxdata[tid] = std::numeric_limits<double>::min();
    }
    __syncthreads();
    const auto sh_limit = _MIN(n, block_size);
#define stride_reduce_suminmax(block_low_)                  \
        if (block_size >= block_low_) {                     \
            constexpr unsigned stride2 = block_low_ / 2;    \
            const auto tid_stride2 = tid + stride2;         \
            if (tid < stride2 && tid_stride2 < sh_limit)           \
                _SMM_OP(sumdata[tid], mindata[tid], maxdata[tid], sumdata[tid_stride2], mindata[tid_stride2], maxdata[tid_stride2]); \
            __syncthreads();                                \
        }

    stride_reduce_suminmax(1024);
    stride_reduce_suminmax(512);
    stride_reduce_suminmax(256);
    stride_reduce_suminmax(128);
    if (tid >= 32) return;
    warp_reduce_suminmax<block_size>(sumdata, mindata, maxdata, tid, sh_limit);
    if (tid) return;
    d_result_sum[blockIdx.x] = sumdata[0];
    d_result_min[blockIdx.x] = mindata[0];
    d_result_max[blockIdx.x] = maxdata[0];
}

template<const unsigned block_size> __global__ void
G_suminmax(const double *__restrict__ d_in_sum, const double *__restrict__ d_in_min, const double *__restrict__ d_in_max,
           double *__restrict__ d_result_sum, double *__restrict__ d_result_min, double *__restrict__ d_result_max,
           const size_t n)
{
    __shared__ double sumdata[block_size], mindata[block_size], maxdata[block_size];
    if (tid < n) {
        sumdata[tid] = d_in_sum[tid];
        mindata[tid] = d_in_min[tid];
        maxdata[tid] = d_in_max[tid];
    } else {
        sumdata[tid] = 0;
        mindata[tid] = std::numeric_limits<double>::max();
        maxdata[tid] = std::numeric_limits<double>::min();
    }
    __syncthreads();
    const auto sh_limit = _MIN(n, block_size);
    stride_reduce_suminmax(1024);
    stride_reduce_suminmax(512);
    stride_reduce_suminmax(256);
    stride_reduce_suminmax(128);
    if (tid >= 32) return;
    warp_reduce_suminmax<block_size>(sumdata, mindata, maxdata, tid, sh_limit);
    if (tid) return;
    d_result_sum[0] = sumdata[0];
    d_result_min[0] = mindata[0];
    d_result_max[0] = maxdata[0];
}

std::tuple<double, double, double> suminmax(const double *d_in, const size_t n, const hipStream_t &stm)
{
    double sum, min, max;
    const auto clamped_n = clamp_n(n);
    const auto grid_len = CUDA_BLOCKS(clamped_n);
    const auto threads = CUDA_THREADS(clamped_n);
    const auto grid_size = grid_len * sizeof(double);
    double *d_sum, *d_min, *d_max;
    cu_errchk(hipMallocAsync((void **) &d_sum, sizeof(double), stm));
    cu_errchk(hipMallocAsync((void **) &d_min, sizeof(double), stm));
    cu_errchk(hipMallocAsync((void **) &d_max, sizeof(double), stm));
    if (grid_len > 1) {
        double *d_block_sums, *d_block_mins, *d_block_maxs;
        cu_errchk(hipMallocAsync((void **) &d_block_sums, grid_size, stm));
        cu_errchk(hipMallocAsync((void **) &d_block_mins, grid_size, stm));
        cu_errchk(hipMallocAsync((void **) &d_block_maxs, grid_size, stm));
        G_suminmax < common::C_cu_block_size ><<<grid_len, threads, 0, stm>>>(d_in, d_block_sums, d_block_mins, d_block_maxs, n);
        G_suminmax < common::C_cu_block_size ><<<1, grid_len, 0, stm>>>(d_block_sums, d_block_mins, d_block_maxs, d_sum, d_min, d_max, grid_len);
        cu_errchk(hipFreeAsync(d_block_sums, stm));
        cu_errchk(hipFreeAsync(d_block_mins, stm));
        cu_errchk(hipFreeAsync(d_block_maxs, stm));
    } else
        G_suminmax < common::C_cu_block_size ><<<grid_len, threads, 0, stm>>>(d_in, d_sum, d_min, d_max, n);
    cu_errchk(hipMemcpyAsync(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipMemcpyAsync(&min, d_min, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipMemcpyAsync(&max, d_max, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipFreeAsync(d_sum, stm));
    cu_errchk(hipFreeAsync(d_min, stm));
    cu_errchk(hipFreeAsync(d_max, stm));
    cu_errchk(hipStreamSynchronize(stm));
    return {sum, min, max};
}

std::tuple<double, double, double> meanminmax(const double *d_in, const size_t n, const hipStream_t &stm)
{
    const auto [sum, min, max] = suminmax(d_in, n, stm);
    return {sum / double(n), min, max};
}


template<const unsigned block_size> __device__ inline void
warp_reduce_dist(volatile double *sh_dist, const unsigned ix, const unsigned n)
{
#define _DO_WARP_REDUCE_DIST(N) {               \
    if (block_size >= (N)) {                    \
        const unsigned ix_N_2 = ix + (N) / 2;   \
        if (ix_N_2 < n)                         \
            sh_dist[ix] += sh_dist[ix_N_2];     \
        } }

    _DO_WARP_REDUCE_DIST(64);
    _DO_WARP_REDUCE_DIST(32);
    _DO_WARP_REDUCE_DIST(16);
    _DO_WARP_REDUCE_DIST(8);
    _DO_WARP_REDUCE_DIST(4);
    _DO_WARP_REDUCE_DIST(2);
}

template<const unsigned block_size> __global__ void G_dist_unscaled(
        double *__restrict__ d_mae, const double *__restrict__ d_labels, const double *__restrict__ d_predictions, const size_t n)
{
    __shared__ double sh_dist[block_size];
    auto i = blockIdx.x * block_size + tid;
    if (i < n) {
        sh_dist[tid] = fabs(d_labels[i] - d_predictions[i]);
        const auto stride1 = blockDim.x * gridDim.x;
#ifdef PRODUCTION_BUILD
#pragma unroll
#endif
        for (i += stride1; i < n; i += stride1)
            sh_dist[tid] += fabs(d_labels[i] - d_predictions[i]);
    } else
        sh_dist[tid] = 0;

    __syncthreads();

#define stride_reduce_dist(block_low_)                                  \
        if (block_size >= block_low_) {                                 \
            constexpr unsigned stride2 = block_low_ / 2;                \
            const auto tid_stride2 = tid + stride2;                     \
            if (tid < stride2 && tid_stride2 < n)                       \
                sh_dist[tid] += sh_dist[tid_stride2];                   \
            __syncthreads();                                            \
        }

    stride_reduce_dist(1024);
    stride_reduce_dist(512);
    stride_reduce_dist(256);
    stride_reduce_dist(128);
    if (tid >= 32) return;
    warp_reduce_dist<block_size>(sh_dist, tid, n);
    if (tid) return;
    d_mae[blockIdx.x] = sh_dist[0];
}

double
unscaled_distance(const double *d_labels, const double *d_predictions, const double sf, const size_t n, const hipStream_t stm)
{
    const auto clamped_n = clamp_n(n);
    const auto grid_len = CUDA_BLOCKS(clamped_n);
    double *d_block_sums;
    cu_errchk(hipMallocAsync((void **) &d_block_sums, grid_len * sizeof(double), stm));
    G_dist_unscaled < common::C_cu_block_size ><<<CUDA_THREADS_BLOCKS(clamped_n), 0, stm>>>(d_block_sums, d_labels, d_predictions, n);
    double mae;
    if (grid_len > 1)
        mae = thrust::async::reduce(thrust::cuda::par.on(stm), d_block_sums, d_block_sums + grid_len, double(0), thrust::plus<double>()).get();
    else cu_errchk(hipMemcpyAsync(&mae, d_block_sums, sizeof(double), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipFreeAsync(d_block_sums, stm));
    cu_errchk(hipStreamSynchronize(stm));
    // LOG4_TRACE("Length " << n << ", scaling factor " << sf << ", DC offset " << dc << ", grid len " << grid_len << ", threads " << CUDA_THREADS(clamped_n) << ", sum dist " << res);
    return sf * mae / double(n);
}


double max(const double *d_in, const size_t n, const hipStream_t stm)
{
    return thrust::async::reduce(thrust::cuda::par.on(stm), d_in, d_in + n, std::numeric_limits<double>::min(), thrust::maximum<double>()).get();
}

double min(const double *d_in, const size_t n, const hipStream_t stm)
{
    return thrust::async::reduce(thrust::cuda::par.on(stm), d_in, d_in + n, std::numeric_limits<double>::max(), thrust::minimum<double>()).get();
}

double mean(const double *d_in, const size_t n, const hipStream_t &stm)
{
    return sum(d_in, n, stm) / double(n);
}

double sum(const double *d_in, const size_t n, const hipStream_t &stm)
{
#if 0
    size_t npp_buffer_size;
    if (nppGetStream() != stm) np_errchk(nppSetStream(stm));
    np_errchk(nppsSumGetBufferSize_64f(n, &npp_buffer_size));
    // auto npp_mean_buf = nppsMalloc_8u(npp_buffer_size);
    Npp8u *npp_sum_buf;
    Npp64f *dres;
    cu_errchk(hipMallocAsync((void **)&npp_sum_buf, npp_buffer_size, stm));
    cu_errchk(hipMallocAsync((void **)&dres, sizeof(*dres), stm));
    assert(npp_sum_buf != nullptr);
    np_errchk(nppsSum_64f(d_in, n, dres, npp_sum_buf));
    double res;
    cu_errchk(hipMemcpyAsync(&res, dres, sizeof(*dres), hipMemcpyDeviceToHost, stm));
    cu_errchk(hipStreamSynchronize(stm));
    cu_errchk(hipFreeAsync(npp_sum_buf, stm));
    cu_errchk(hipFreeAsync(dres, stm));
    return res;
#else
    return thrust::async::reduce(thrust::cuda::par.on(stm), d_in, d_in + n, double(0), thrust::plus<double>()).get();
#endif
}


__global__ void G_sqrt_add(double *__restrict__ input, const double a, const size_t N)
{
    CUDA_STRIDED_FOR_i(N)input[i] = sqrt(input[i] + a);
}

__global__ void G_matmul_inplace(const double *__restrict__ input, double *__restrict__ output, const size_t N)
{
    CUDA_STRIDED_FOR_i(N)output[i] *= input[i];
}

__global__ void G_eq_matmul(const double *__restrict__ input1, const double *__restrict__ input2, double *__restrict__ output, const size_t N)
{
    CUDA_STRIDED_FOR_i(N)output[i] = input1[i] * input2[i];
}

__global__ void G_abs_subtract(const double *__restrict__ input1, double *__restrict__ input2, const size_t N)
{
    CUDA_STRIDED_FOR_i(N)input2[i] = std::abs(input1[i] - input2[i]);
}

#if 0 // Not used
void solve_irwls(const arma::mat &K_epsco, const arma::mat &K, const arma::mat &rhs, arma::mat &solved, const size_t iters, const size_t gpu_phy_id)
{
    const bool psd = false;
#ifdef USE_MAGMA
    auto [magma_queue, d_K, d_rhs, d_x, d_tmpd, d_tmpf, piv] = solvers::init_magma_solver(K.n_rows, rhs.n_cols, psd, gpu_phy_id);
#else
    auto [cusolverH, d_K, d_rhs, d_tmpd, d_piv, d_devinfo] = solvers::init_cusolver(gpu_phy_id, K.n_rows, rhs.n_cols);
#endif
    if (arma::size(solved) != arma::size(rhs)) solved = rhs; // TODO Should be set_size

#ifdef USE_MAGMA
    solvers::iter_magma_solve(K_epsco.n_rows, rhs.n_cols, K_epsco.mem, rhs.mem, solved.memptr(), magma_queue, d_K, d_rhs, d_x, d_tmpd, d_tmpf, psd, gpu_phy_id);
#else
    solvers::dyn_gpu_solve(gpu_phy_id, K_epsco.n_rows, rhs.n_cols, K_epsco.mem, rhs.mem, solved.memptr(), cusolverH, d_K, d_rhs, d_tmpd, d_piv, d_devinfo);
#endif
    auto best_sae = std::numeric_limits<double>::infinity();
    arma::mat best_solution = solved;
    size_t best_iter = 0;
#pragma omp unroll
    for (size_t i = 1; i < iters; ++i) {
        const arma::mat error_mat = arma::abs(K * solved - rhs);
        const double this_sae = arma::accu(error_mat);
        if (this_sae < best_sae) {
            LOG4_TRACE("IRWLS iteration " << i << ", SAE " << this_sae << ", kernel dimensions " << arma::size(K) << ", best SAE " << best_sae);
            best_sae = this_sae;
            best_solution = solved;
            best_iter = i;
        }
        const arma::mat mult = arma::sqrt(error_mat + common::C_itersolve_delta / (double(i) * common::C_itersolve_range / double(iters)));
        const arma::mat left = (mult * arma::ones(mult.n_cols, K.n_cols)) % K_epsco;
        const arma::mat right = rhs % mult;
#ifdef USE_MAGMA
        solvers::iter_magma_solve(left.n_rows, right.n_cols, left.mem, right.mem, solved.memptr(), magma_queue, d_K, d_rhs, d_x, d_tmpd, d_tmpf, psd, gpu_phy_id);
#else
        solvers::dyn_gpu_solve(gpu_phy_id, left.n_rows, right.n_cols, left.mem, right.mem, solved.memptr(), cusolverH, d_K, d_rhs, d_tmpd, d_piv, d_devinfo);
#endif
    }

#ifdef USE_MAGMA
    solvers::uninit_magma_solver(magma_queue, d_K, d_rhs, d_x, d_tmpd, d_tmpf, piv, gpu_phy_id);
#else
    solvers::uninit_cusolver(gpu_phy_id, cusolverH, d_K, d_rhs, d_tmpd, d_piv, d_devinfo);
#endif
    LOG4_DEBUG("IRWLS best iteration " << best_iter << ", MAE " << best_sae / double(solved.n_elem) << ", kernel dimensions " << arma::size(K) <<
                                       ", delta " << common::C_itersolve_delta << ", range " << common::C_itersolve_range << ", solution " << arma::size(solved));
    solved = best_solution;
}
#endif

// Namespaces
}
}
