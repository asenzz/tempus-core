#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include "common/defines.h"
#include "cuda_path.cuh"
#include "common/cuda_util.cuh"
#include "common/gpu_handler.hpp"
#include "common/constants.hpp"
#include "onlinesvr.hpp"
#include "model/SVRParameters.hpp"
#include "cuqrsolve.cuh"
#include "common/constants.hpp"

// TODO Reimplement properly the path kernel (or a better choice eg. NTK) according to https://www.csc.kth.se/~fpokorny/static/publications/baisero2013a.pdf

// #define HIFI_PATH // Actually lowers precision when tuning parameters, so keep off for now

#define blockXX(i, j) (X[(i) * rows + (j)])
#define blockYY(i, j) (Y[(i) * rows + (j)])

#define tx threadIdx.x
#define ty threadIdx.y

namespace svr {
namespace kernel::path {


__device__ __forceinline__ double
do_product_sum(const uint32_t rows, const uint32_t lag, const uint32_t dim, const uint32_t lag_TILE_WIDTH, const double lambda, const double tau, CRPTRd X,
               CRPTRd Y, double power_mult[32], double ta[32][32], double tam1[32][32], double tb[32][32], double tbm1[32][32], const uint32_t kk,
               const uint32_t mm, const bool kk_X, const bool mm_Y, const bool do_matrix_product_sum)
{
    double matrix_prod_sum = 0;
UNROLL()
    for (uint32_t jA = 0; jA < dim; ++jA) {
        const auto jA_lag = jA * lag;
UNROLL()
        for (uint32_t kk_internal_big = 0; kk_internal_big < lag_TILE_WIDTH; ++kk_internal_big) {
            const auto kk_internal_big_TILE_WIDTH = kk_internal_big * common::C_cu_tile_width;
            const auto ty_kk_internal_big_TILE_WIDTH = ty + kk_internal_big_TILE_WIDTH;
            if (ty_kk_internal_big_TILE_WIDTH < lag) {
                if (!tx) power_mult[ty] = pow(1. / double(lag - ty_kk_internal_big_TILE_WIDTH), lambda);
                if (kk_X) {
                    ta[tx][ty] = blockXX(kk, ty_kk_internal_big_TILE_WIDTH + jA_lag);
                    if (ty_kk_internal_big_TILE_WIDTH) tam1[tx][ty] = ta[tx][ty] - blockXX(kk, ty_kk_internal_big_TILE_WIDTH + jA_lag - 1);
                }
            }

            const auto tx_kk_internal_big_TILE_WIDTH = tx + kk_internal_big_TILE_WIDTH;
            if (mm_Y && tx_kk_internal_big_TILE_WIDTH < lag) {
                tb[ty][tx] = blockYY(mm, tx_kk_internal_big_TILE_WIDTH + jA_lag);
                if (tx_kk_internal_big_TILE_WIDTH) tbm1[ty][tx] = tb[ty][tx] - blockYY(mm, tx_kk_internal_big_TILE_WIDTH + jA_lag - 1);
            }
            __syncthreads();

            if (do_matrix_product_sum)
UNROLL(common::C_cu_tile_width)
                for (uint32_t kk_internal_small = 0; kk_internal_small < common::C_cu_tile_width; ++kk_internal_small) {
                    const auto kk_internal = kk_internal_small + kk_internal_big_TILE_WIDTH;
                    if (kk_internal >= lag) continue;
#ifdef HIFI_PATH
                    matrix_prod_sum += (DIST(ta[tx][kk_internal_small] - tb[ty][kk_internal_small]) +
                            (kk_internal ? (C_kernel_path_tau * DIST(tam1[tx][kk_internal_small] - tbm1[ty][kk_internal_small])) : 0.)) *
                                    power_mult[kk_internal_small] / double(dim);
#else
                    matrix_prod_sum += (DIST(ta[tx][kk_internal_small] - tb[ty][kk_internal_small]) +
                            (kk_internal ? (C_kernel_path_tau * DIST(tam1[tx][kk_internal_small] - tbm1[ty][kk_internal_small])) : 0.)) *
                                    power_mult[kk_internal_small];
#endif
                }
            __syncthreads();
        }
    }
    return matrix_prod_sum;
}

__global__  void
G_kernel_xy(const uint32_t X_cols, const uint32_t Y_cols, const uint32_t rows, const uint32_t lag, const uint32_t dim, const uint32_t lag_TILE_WIDTH, const double lambda,
            const double tau, const double *const X, const double *const Y, double *Z)
{
    if (blockIdx.x * blockDim.x >= X_cols || blockIdx.y * blockDim.y >= Y_cols) return;

    __shared__ double power_mult[common::C_cu_tile_width];
    __shared__ double ta[common::C_cu_tile_width][common::C_cu_tile_width];
    __shared__ double tam1[common::C_cu_tile_width][common::C_cu_tile_width]; // for index-1
    __shared__ double tb[common::C_cu_tile_width][common::C_cu_tile_width];
    __shared__ double tbm1[common::C_cu_tile_width][common::C_cu_tile_width]; // for index-1

    const auto kk = threadIdx.x + blockIdx.x * blockDim.x;
    const auto mm = threadIdx.y + blockIdx.y * blockDim.y;
    const bool kk_X = kk < X_cols;
    const bool mm_Y = mm < Y_cols;
    const auto do_matrix_product_sum = mm_Y && kk_X;
    const auto matrix_prod_sum = do_product_sum(
            rows, lag, dim, lag_TILE_WIDTH, lambda, tau, X, Y, power_mult, ta, tam1, tb, tbm1, kk, mm, kk_X, mm_Y, do_matrix_product_sum);
    if (do_matrix_product_sum)
#ifdef HIFI_PATH
        Z[kk * Y_cols + mm] = matrix_prod_sum;
#else
        Z[kk * Y_cols + mm] = matrix_prod_sum / dim;
#endif
}


__global__  void
G_kernel_xy(const uint32_t X_cols, const uint32_t Y_cols, const uint32_t rows, const uint32_t lag, const uint32_t dim, const uint32_t lag_TILE_WIDTH, const double lambda,
            const double tau, const double gamma, const double *const X, const double *const Y, double *Z)
{
    if (blockIdx.x * blockDim.x >= X_cols || blockIdx.y * blockDim.y >= Y_cols) return;

    __shared__ double power_mult[common::C_cu_tile_width];
    __shared__ double ta[common::C_cu_tile_width][common::C_cu_tile_width];
    __shared__ double tam1[common::C_cu_tile_width][common::C_cu_tile_width];
    __shared__ double tb[common::C_cu_tile_width][common::C_cu_tile_width];
    __shared__ double tbm1[common::C_cu_tile_width][common::C_cu_tile_width];

    const auto kk = threadIdx.x + blockIdx.x * blockDim.x;
    const auto mm = threadIdx.y + blockIdx.y * blockDim.y;
    const bool kk_X = kk < X_cols;
    const bool mm_Y = mm < Y_cols;
    const auto do_matrix_product_sum = mm_Y && kk_X;
    const auto matrix_prod_sum = do_product_sum(rows, lag, dim, lag_TILE_WIDTH, lambda, 0, X, Y, power_mult, ta, tam1, tb, tbm1, kk, mm, kk_X, mm_Y, do_matrix_product_sum);
    if (do_matrix_product_sum)
#ifdef HIFI_PATH
        Z[kk * Y_cols + mm] = matrix_prod_sum;
#else
        Z[kk * Y_cols + mm] = matrix_prod_sum / dim / gamma;
#endif
}

__global__ void G_threshold(RPTR(double) Z, const uint32_t len, const double threshold)
{
    const auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len && abs(Z[i]) < threshold) Z[i] = 0;
}

void cu_threshold(RPTR(double) v, const uint32_t n, const hipStream_t custream)
{
    return;

    const auto meanabs = solvers::meanabs(v, n, custream);
    G_threshold<<<CU_BLOCKS_THREADS(n), 0, custream>>>(v, n, meanabs / 2);
}

void cu_distances_xx(const uint32_t cols, const uint32_t rows, const uint32_t lag, const double lambda, const double tau, CRPTR(double) X, RPTR(double) Z)
{
    assert(rows % lag == 0);
    const uint32_t X_size = cols * rows * sizeof(double);
    const auto Z_len = cols * cols;
    const uint32_t Z_size = Z_len * sizeof(double);
    double *d_Z, *d_X;
    CTX_CUSTREAM;
    cu_errchk(hipMallocAsync(&d_X, X_size, custream));
    cu_errchk(hipMemcpyAsync(d_X, X, X_size, hipMemcpyHostToDevice, custream));
    cu_errchk(hipMallocAsync(&d_Z, Z_size, custream));
    G_kernel_xy<<<CU_BLOCKS_THREADS_2D(cols), 0, custream>>>(cols, cols, rows, lag, rows / lag, CDIVI(lag, common::C_cu_tile_width), lambda, 0, d_X, d_X, d_Z);
    cu_threshold(d_Z, Z_len, custream);
    cu_errchk(hipFreeAsync(d_X, custream));
    cu_errchk(hipMemcpyAsync(Z, d_Z, Z_size, hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_Z, custream));
    cusyndestroy(custream);
}


void
cu_distances_xy(const uint32_t X_cols, const uint32_t Xy_cols, const uint32_t rows, const uint32_t lag, const double lambda, const double tau, CRPTRd X, CRPTRd Xy, RPTR(double) Z)
{
    const uint32_t X_size = X_cols * lag * sizeof(double);
    const uint32_t Xy_size = Xy_cols * lag * sizeof(double);
    const auto Z_len = X_cols * Xy_cols;
    const uint32_t Z_size = Z_len * sizeof(double);
    double *d_X, *d_Xy, *d_Z;
    CTX_CUSTREAM;
    cu_errchk(hipMallocAsync(&d_X, X_size, custream));
    cu_errchk(hipMemcpyAsync(d_X, X, X_size, hipMemcpyHostToDevice, custream));
    cu_errchk(hipMallocAsync(&d_Xy, Xy_size, custream));
    cu_errchk(hipMemcpyAsync(d_Xy, Xy, Xy_size, hipMemcpyHostToDevice, custream));
    cu_errchk(hipMallocAsync(&d_Z, Z_size, custream));
    G_kernel_xy<<<CU_BLOCKS_THREADS_2D(X_cols), 0, custream>>>(X_cols, Xy_cols, lag, rows, rows / lag, CDIVI(lag, common::C_cu_tile_width), lambda, 0, d_X, d_Xy, d_Z);
    cu_threshold(d_Z, Z_len, custream);
    cu_errchk(hipFreeAsync(d_X, custream));
    cu_errchk(hipFreeAsync(d_Xy, custream));
    cu_errchk(hipMemcpyAsync(Z, d_Z, Z_size, hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_Z, custream));
    cusyndestroy(custream);
}


void cu_kernel_xx(const uint32_t cols, const uint32_t rows, const uint32_t lag, const double lambda, const double tau, const double gamma, CRPTRd X, RPTR(double) K)
{
    LOG4_THROW("Kills precision!");
    const auto K_len = cols * cols;
    const uint32_t K_size = K_len * sizeof(double);
    double *d_K;
    CTX4_CUSTREAM;
    const auto d_X = cumallocopy(X, custream, rows * cols);
    cu_errchk(hipMallocAsync(&d_K, K_size, custream));
    G_kernel_xy<<<CU_BLOCKS_THREADS_2D(cols), 0, custream>>>(cols, cols, rows, lag, rows / lag, CDIVI(lag, common::C_cu_tile_width), lambda, 0, gamma, d_X, d_X, d_K);
    cu_errchk(hipFreeAsync(d_X, custream));
    cufreecopy(K, d_K, custream, K_len);
    cusyndestroy(custream);
}


void cu_kernel_xy(const uint32_t X_cols, const uint32_t Xy_cols, const uint32_t rows, const uint32_t lag, const double lambda, const double gamma,
                  CPTRd X, CPTRd Xy, RPTR(double) K)
{
    LOG4_THROW("Kills precision!");
    const auto K_len = X_cols * Xy_cols;
    const auto K_size = K_len * sizeof(double);
    double *d_K;
    CTX4_CUSTREAM;
    const auto d_X = cumallocopy(X, custream, X_cols * rows);
    const auto d_Xy = cumallocopy(Xy, custream, Xy_cols * rows);
    cu_errchk(hipMallocAsync(&d_K, K_size, custream));
    G_kernel_xy<<<CU_BLOCKS_THREADS_2D(X_cols), 0, custream>>>(X_cols, Xy_cols, rows, lag, rows / lag, CDIVI(lag, common::C_cu_tile_width), lambda, 0, gamma, d_X, d_Xy, d_K);
    cu_errchk(hipFreeAsync(d_X, custream));
    cu_errchk(hipFreeAsync(d_Xy, custream));
    cu_errchk(hipMemcpyAsync(K, d_K, K_size, hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_K, custream));
    cusyndestroy(custream);
}


}
}
