#include "hip/hip_runtime.h"
//
// Created by zarko on 10/3/22.
//
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <queue>
#include <iomanip>
#include <thrust/device_vector.h>
#include <thrust/logical.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <cmath>
#include <vector>
#include <algorithm>
#include <boost/math/special_functions/bessel.hpp>
#include "pprune.hpp"
#include "common/compatibility.hpp"
#include "oemd_coefficient_search.hpp"
#include "online_emd.hpp"
#include "../../SVRCommon/include/common/cuda_util.cuh"
#include "util/time_utils.hpp"
#include "firefly.hpp"
#include "common/logging.hpp"
#include "cuqrsolve.cuh"
#include "align_features.cuh"
#include "onlinesvr.hpp"
#include "ModelService.hpp"
#include "appcontext.hpp"

// #define USE_FIREFLY // else use BITEOPT

namespace svr {
namespace oemd {


namespace {

constexpr unsigned C_column_interleave = datamodel::C_features_superset_coef;

constexpr unsigned C_quantisation_interleave = 3;
#ifdef PRODUCTION_BUILD
constexpr unsigned unroll_ct = CDIVI(datamodel::C_default_svrparam_lag_count * datamodel::C_features_superset_coef / C_column_interleave, 10);
#endif
}

bool cu_fix_mask(double *const d_mask, const unsigned mask_len, const hipStream_t custream)
{
    const auto mask_sum = solvers::sum(d_mask, mask_len, custream);
    if (mask_sum == 0.) {
        LOG4_WARN("Zero mask sum.");
        return false;
    }
    datamodel::G_div_inplace<<<CU_BLOCKS_THREADS(mask_len), 0, custream>>>(d_mask, mask_sum, mask_len);
    return true;
}

void fix_mask(CPTR(double) h_in, double *const h_out, const unsigned mask_len, const hipStream_t custream)
{
    auto d_in = cumallocopy(h_in, mask_len, hipMemcpyHostToDevice, custream);
    cu_fix_mask(d_in, mask_len, custream);
    cufreecopy(h_out, d_in, custream, mask_len);
}

// This function should be equivalent to align_features.cu:cu_align_features
template<const unsigned block_size> __global__ void G_autocorrelation_sum(
        RPTR(double) d_sum, CRPTR(double) x, CRPTR(double) y, const unsigned n_min, const unsigned qt, const unsigned n_qt)
{
    static __shared__ double sh_dist[block_size];
    sh_dist[tid] = 0;
    CU_STRIDED_FOR_i(n_qt) {
        double y_qi = 0;
        const auto to_q = (i + 1) * qt;
        UNROLL()
        for (unsigned qi = i * qt; qi < to_q; ++qi) y_qi += y[qi];
        y_qi /= qt;
        sh_dist[tid] += fabs(x[i] - y_qi) / (fabs(x[i]) + fabs(y_qi));
    }

    __syncthreads();

#define stride_reduce_dist(block_low_)                                                  \
        if (block_size >= block_low_) {                                                 \
            constexpr unsigned stride2 = block_low_ / 2;                                \
            const auto tid_stride2 = tid + stride2;                                     \
            if (tid < stride2 && tid_stride2 < n_min)                                   \
                sh_dist[tid] += sh_dist[tid_stride2];                                   \
            __syncthreads();                                                            \
        }

    stride_reduce_dist(1024);
    stride_reduce_dist(512);
    stride_reduce_dist(256);
    stride_reduce_dist(128);
    if (tid >= 32) return;
    warp_reduce_sum<block_size>(sh_dist, tid, n_min);
    if (tid) return;
    atomicAdd(d_sum, *sh_dist);
}

template<const unsigned block_size> __global__ void G_autocorrelation_block(
        RPTR(double) d_sum, CRPTR(double) x, CRPTR(double) y, const unsigned n, const unsigned n_min, const float st)
{
//    constexpr float sk = 1; // Skip is disabled for now
    static __shared__ double sh_dist[block_size];
    sh_dist[tid] = 0;
    CU_STRIDED_FOR_i(n) {
        const auto y_i = y[STRETCHSKIP_(i)];
        sh_dist[tid] += fabs(x[i] - y_i) / (fabs(x[i]) + fabs(y_i));
    }

    __syncthreads();

    stride_reduce_dist(1024);
    stride_reduce_dist(512);
    stride_reduce_dist(256);
    stride_reduce_dist(128);
    if (tid >= 32) return;
    warp_reduce_sum<block_size>(sh_dist, tid, n_min);
    if (tid) return;
    d_sum[blockIdx.x] = *sh_dist;
}

__global__ void G_autocorr_driver(CRPTR(double) in, CRPTR(double) in_n, CRPTR(unsigned) offsets, RPTR(double) res, const unsigned n_offsets)
{
    CU_STRIDED_FOR_i(n_offsets) {
        const auto off = offsets[i];
        const auto blocks = CU_BLOCKS(clamp_n(off));
        const auto threads = CU_THREADS(off);
        auto res_i = (double *) malloc(blocks * sizeof(double));
        UNROLL()
        for (float st = 1; st > C_stretch_limit; st *= C_stretch_multiplier) {
            G_autocorrelation_block<common::C_cu_block_size><<<blocks, threads>>>(res_i, in, in_n - off, off, threads, st);
            const auto this_res = thrust::reduce(thrust::seq, res_i, res_i + blocks) / off;
            if (this_res < res[i]) res[i] = this_res;
        }
        free(res_i);
    }
}

double autocorrelation_n(CPTR(double) d_in, const unsigned n, const std::vector<unsigned> &offsets, const hipStream_t &stm)
{
    const auto d_offsets = cumallocopy(offsets, stm);
    const unsigned n_offsets = offsets.size();
    double *d_res;
    cu_errchk(hipMallocAsync((void **) &d_res, n_offsets * sizeof(*d_res), stm));
    G_autocorr_driver<<<CU_BLOCKS_THREADS(n_offsets), 0, stm>>>(d_in, d_in + n, d_offsets, d_res, n_offsets);
    cu_errchk(hipFreeAsync(d_offsets, stm));
    thrust::sort(thrust::cuda::par.on(stm), d_res, d_res + n_offsets);
    const auto n_offsets_2 = n_offsets;
    const auto res = solvers::sum(d_res, n_offsets_2, stm);
    cu_errchk(hipFreeAsync(d_res, stm));
    return res / n_offsets_2;
}

__global__ void
G_multiply_complex(
        const double input_len_div,
        const unsigned fft_len,
        CRPTR(hipfftDoubleComplex) multiplier,
        hipfftDoubleComplex *__restrict__ output)
{
    hipfftDoubleComplex new_output;
    CU_STRIDED_FOR_i(fft_len) {
        new_output.x = output[i].x * multiplier[i].x - output[i].y * multiplier[i].y;
        new_output.y = output[i].x * multiplier[i].y + output[i].y * multiplier[i].x;
        output[i].x = new_output.x / input_len_div; // because of inverse FFT
        output[i].y = new_output.y / input_len_div;
    }
}

__global__ void G_vec_power_inplace(
        hipfftDoubleComplex *__restrict__ x,
        const unsigned x_size_2_1,
        const unsigned siftings)
{
    const auto ix = blockIdx.x * blockDim.x + tid;
    const auto stride = blockDim.x * gridDim.x;
    double px_out, py_out, px, py;
    UNROLL()
    for (auto j = ix; j < x_size_2_1; j += stride) {
        px = 1. - x[j].x;
        py = -x[j].y;
        UNROLL()
        for (unsigned i = 1; i < siftings; i++) {
            px_out = px * (1. - x[j].x) - py * (-x[j].y);
            py_out = px * (-x[j].y) + py * (1. - x[j].x);
            px = px_out;
            py = py_out;
        }
        x[j].x = px;
        x[j].y = py;
    }
}

__global__ void
G_vec_power(
        CRPTR(hipfftDoubleComplex) x,
        hipfftDoubleComplex *__restrict__ y,
        const unsigned n,
        const unsigned siftings)
{
    double px, py;
    CU_STRIDED_FOR_i(n) {
        px = 1. - x[i].x;
        py = -x[i].y;
        for (unsigned j = 1; j < siftings; ++j) {
            px = px * (1. - x[j].x) - py * (-x[j].y);
            py = px * (-x[j].y) + py * (1. - x[j].x);
        }
        y[i].x = px;
        y[i].y = py;
    }
}

__global__ void G_gpu_multiply_smooth(
        const unsigned input_size,
        const double coeff,
        hipfftDoubleComplex *__restrict__ output)
{
    CU_STRIDED_FOR_i(input_size / 2 + 1) {
        const double mult = exp(-coeff * double(i) / double(input_size));
        output[i].x *= mult;
        output[i].y *= mult;
    }
}


__global__ void G_vec_sift(
        const unsigned fft_size,
        const unsigned siftings,
        const hipfftDoubleComplex *__restrict__ x,
        hipfftDoubleComplex *__restrict__ imf,
        hipfftDoubleComplex *__restrict__ rem)
{
    double px, py;
    CU_STRIDED_FOR_i(fft_size) {
        px = 1. - x[i].x;
        py = -x[i].y;
        UNROLL()
        for (unsigned j = 1; j < siftings; ++j) {
            px = px * (1. - x[i].x) - py * (-x[i].y);
            py = px * (-x[i].y) + py * (1. - x[i].x);
        }
        imf[i].x = px;
        imf[i].y = py;
        rem[i].x = 1. - px;
        rem[i].y = -py;
    }
}

__global__ void G_sum_expanded(
        RPTR(double) d_sum_imf, RPTR(double) d_sum_rem, RPTR(double) d_sum_corr, CRPTR(double) d_imf_mask, CRPTR(double) d_rem_mask,
        const unsigned expand_size, CRPTR(double) d_global_sift_matrix)
{
    const double expand_size_2 = expand_size * expand_size;

    __shared__ double _sh_sum_imf[common::C_cu_block_size];
    __shared__ double _sh_sum_rem[common::C_cu_block_size];
    __shared__ double _sh_sum_corr[common::C_cu_block_size];
    _sh_sum_imf[tid] = 0;
    _sh_sum_rem[tid] = 0;
    _sh_sum_corr[tid] = 0;
    CU_STRIDED_FOR_i(expand_size) {
        double sum1 = 0, sum2 = 0;
        UNROLL()
        for (int j = 0; j < expand_size; ++j) {
            const auto abs_i_j = labs(i - j);
            sum1 += d_global_sift_matrix[abs_i_j] * d_imf_mask[j];
            sum2 += d_global_sift_matrix[abs_i_j] * d_rem_mask[j];
        }
        _sh_sum_imf[tid] += sum1 * d_imf_mask[i] / expand_size_2;
        _sh_sum_rem[tid] += sum2 * d_rem_mask[i] / expand_size_2;
        _sh_sum_corr[tid] += sum1 * d_rem_mask[i] / expand_size_2;
    }
    __syncthreads();

    UNROLL()
    for (auto size = common::C_cu_block_size / 2; size > 0; size /= 2) { // uniform
        if (tid >= size) continue;
        _sh_sum_imf[tid] += _sh_sum_imf[tid + size];
        _sh_sum_rem[tid] += _sh_sum_rem[tid + size];
        _sh_sum_corr[tid] += _sh_sum_corr[tid + size];
        __syncthreads();
    }
    if (tid) return;
    atomicAdd(d_sum_imf, *_sh_sum_imf);
    atomicAdd(d_sum_rem, *_sh_sum_rem);
    atomicAdd(d_sum_corr, *_sh_sum_corr);
}


void oemd_coefficients_search::transform(
        double *d_values, CPTR(double) d_mask, const unsigned input_len, const unsigned mask_len,
        const unsigned siftings, double *d_temp, const hipStream_t custream) const
{
    auto d_imf = cumallocopy(d_values, input_len, hipMemcpyDeviceToDevice, custream);
    sift(siftings, input_len, mask_len, custream, d_mask, d_imf, d_temp);
    oemd::G_subtract_inplace<<<CU_BLOCKS_THREADS(input_len), 0, custream>>>(d_values, d_imf, input_len);
    cu_errchk(hipFreeAsync(d_imf, custream));
}


std::tuple<double, double, double, double>
oemd_coefficients_search::sift_the_mask(
        const unsigned mask_size,
        const unsigned siftings,
        CPTR(double) d_mask,
        const hipfftHandle plan_sift_forward,
        const hipfftHandle plan_sift_backward,
        CPTR(double) d_expanded_mask,
        const hipfftDoubleComplex *d_expanded_mask_fft,
        CPTR(double) d_global_sift_matrix_ptr,
        const unsigned gpu_id)
{
    cu_errchk(hipSetDevice(gpu_id));
    hipStream_t custream;
    cu_errchk(hipStreamCreate(&custream));
    double sum_full, sum_imf, sum_rem, sum_corr;
    const unsigned expand_size = siftings * mask_size;
    thrust::device_vector<double> d_zm_mask(expand_size);
    thrust::device_vector<double> d_imf_mask(expand_size);
    thrust::device_vector<double> d_rem_mask(expand_size);

    double *d_expanded_mask_ptr = thrust::raw_pointer_cast(d_zm_mask.data());

    online_emd::expand_the_mask(mask_size, expand_size, d_mask, d_expanded_mask_ptr, custream);
    const auto fft_size = common::fft_len(expand_size);
    thrust::device_vector<hipfftDoubleComplex> d_fzm_mask(fft_size);
    thrust::device_vector<hipfftDoubleComplex> d_mask_imf_fft(fft_size);
    thrust::device_vector<hipfftDoubleComplex> d_mask_rem_fft(fft_size);
    // hipfftDoubleComplex *d_expanded_mask_fft = thrust::raw_pointer_cast(d_fzm_mask.data());
    // cf_errchk(hipfftSetStream(plan_sift_forward, custream));
    // cufft_errchk(hipfftExecD2Z(plan_sift_forward, d_expanded_mask_ptr, d_expanded_mask_fft));
    G_vec_sift<<<CU_BLOCKS_THREADS(fft_size), 0, custream>>>(fft_size, siftings, d_expanded_mask_fft, thrust::raw_pointer_cast(d_mask_imf_fft.data()),
                                                             thrust::raw_pointer_cast(d_mask_rem_fft.data()));
    cf_errchk(hipfftSetStream(plan_sift_backward, custream));
    cf_errchk(hipfftExecZ2D(plan_sift_backward, thrust::raw_pointer_cast(d_mask_imf_fft.data()), thrust::raw_pointer_cast(d_imf_mask.data())));
    cf_errchk(hipfftExecZ2D(plan_sift_backward, thrust::raw_pointer_cast(d_mask_rem_fft.data()), thrust::raw_pointer_cast(d_rem_mask.data())));

    double *d_sum_imf, *d_sum_rem, *d_sum_corr;
    cu_errchk(hipMallocAsync((void **) &d_sum_imf, sizeof(double), custream));
    cu_errchk(hipMallocAsync((void **) &d_sum_rem, sizeof(double), custream));
    cu_errchk(hipMallocAsync((void **) &d_sum_corr, sizeof(double), custream));
    G_sum_expanded<<<CU_BLOCKS_THREADS(expand_size), 0, custream>>>(
            d_sum_imf, d_sum_rem, d_sum_corr, thrust::raw_pointer_cast(d_imf_mask.data()), thrust::raw_pointer_cast(d_rem_mask.data()),
            expand_size, d_global_sift_matrix_ptr);
    cu_errchk(hipMemcpyAsync(&sum_imf, d_sum_imf, sizeof(double), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipMemcpyAsync(&sum_rem, d_sum_rem, sizeof(double), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipMemcpyAsync(&sum_corr, d_sum_corr, sizeof(double), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_sum_imf, custream));
    cu_errchk(hipFreeAsync(d_sum_rem, custream));
    cu_errchk(hipFreeAsync(d_sum_corr, custream));
    cu_errchk(hipMemcpyAsync(&sum_full, d_global_sift_matrix_ptr, sizeof(double), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipStreamSynchronize(custream));
    cu_errchk(hipStreamDestroy(custream));
    return {sum_full, sum_imf, sum_rem, sum_corr};
}


__global__ void G_do_quality(
        CRPTR(hipDoubleComplex) mask_fft, const unsigned siftings, const unsigned n, const double coeff, const unsigned end_i, const double mask_fft_coef, double *result)
{
    __shared__ double shared[common::C_cu_block_size];
    constexpr hipDoubleComplex cplx_one{1, 0};
    CU_STRIDED_FOR_i(n) {
        hipDoubleComplex p{1, 0};
        if (i < end_i) {
            const auto zz = make_hipDoubleComplex(1. - mask_fft[i].x, -mask_fft[i].y);
            UNROLL()
            for (unsigned k = 0; k < siftings; ++k) p = hipCmul(p, zz);
            shared[tid] = cunorm(p) + abs(1. - cunorm(hipCsub(cplx_one, p)));
        } else {
            const hipDoubleComplex zz = mask_fft[i];
            UNROLL()
            for (unsigned k = 0; k < siftings; ++k) p = hipCmul(p, zz);
            shared[tid] = i < mask_fft_coef ? cunorm(p) : oemd_coefficients_search::C_smooth_factor * cunorm(p);
        }
        const double norm_zz = cunorm(mask_fft[i]);
        if (norm_zz > 1) shared[tid] += norm_zz;
    }
    __syncthreads();

    const auto sh_limit = _MIN(n, common::C_cu_block_size);
#define stride_reduce_sum(block_low_)                        \
        if (common::C_cu_block_size >= block_low_) {         \
            constexpr unsigned stride2 = block_low_ / 2;     \
            const auto tid_stride2 = tid + stride2;          \
            if (tid < stride2 && tid_stride2 < sh_limit)     \
                shared[tid] += shared[tid_stride2];          \
            __syncthreads();                                 \
        }

    stride_reduce_sum(1024);
    stride_reduce_sum(512);
    stride_reduce_sum(256);
    stride_reduce_sum(128);

    if (tid >= 32) return;
    warp_reduce_sum<common::C_cu_block_size>(shared, tid, sh_limit);

    if (tid) return;
    atomicAdd(result, shared[0]);
}

double oemd_coefficients_search::cu_quality(const hipfftDoubleComplex *mask_fft, const unsigned mask_size, const unsigned siftings, const hipStream_t custream)
{
    const double coeff = mask_size / 250.;
    const unsigned end_i = mask_size * 2. * lambda1 / coeff;
    const auto mask_fft_coef = mask_size * 2. * lambda2 / coeff;
    double result, *d_result = cucalloc<double>(custream);
    G_do_quality<<<CU_BLOCKS_THREADS(mask_size), 0, custream>>>(mask_fft, siftings, mask_size, coeff, end_i, mask_fft_coef, d_result);
    cu_errchk(hipMemcpyAsync(&result, d_result, sizeof(double), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_result, custream));
    cu_errchk(hipStreamSynchronize(custream));
    return result / mask_size;
}


void
oemd_coefficients_search::gauss_smoothen_mask(
        const unsigned mask_size,
        std::vector<double> &mask,
        common::t_drand48_data_ptr buffer,
        hipfftHandle plan_mask_forward,
        hipfftHandle plan_mask_backward,
        const unsigned gpu_id)
{
    const auto full_size = 2 * mask_size;
    cu_errchk(hipSetDevice(gpu_id));
    hipStream_t custream;
    cu_errchk(hipStreamCreate(&custream));
    double *d_mask_zm;
    cu_errchk(hipMallocAsync((void **) &d_mask_zm, full_size, custream));
    hipfftDoubleComplex *d_mask_zm_fft;
    const auto full_fft = common::fft_len(full_size);
    cu_errchk(hipMallocAsync((void **) &d_mask_zm_fft, full_fft, custream));
    cu_errchk(hipMemsetAsync(d_mask_zm + mask_size, 0, mask_size * sizeof(double), custream));
    cu_errchk(hipMemcpyAsync(d_mask_zm, mask.data(), sizeof(double) * mask_size, hipMemcpyKind::hipMemcpyHostToDevice, custream));
    cf_errchk(hipfftSetStream(plan_mask_forward, custream));
    cf_errchk(hipfftExecD2Z(plan_mask_forward, d_mask_zm, d_mask_zm_fft));
    G_gpu_multiply_smooth<<<CU_BLOCKS_THREADS(full_fft), 0, custream>>>(full_size, 5. * -log(common::drander(buffer)), d_mask_zm_fft);
    cf_errchk(hipfftSetStream(plan_mask_backward, custream));
    cf_errchk(hipfftExecZ2D(plan_mask_backward, d_mask_zm_fft, d_mask_zm));
    thrust::transform(thrust::cuda::par.on(custream), d_mask_zm, d_mask_zm + mask_size, d_mask_zm,
    [mask_size]
            __device__(
    const double &iter) -> double{return iter > 0 ? iter / double(mask_size) : 0;} );
    if (mask.size() != full_size) mask.resize(full_size);
    cu_errchk(hipMemcpyAsync(mask.data(), d_mask_zm, full_size * sizeof(*d_mask_zm), hipMemcpyDeviceToHost, custream));
    cu_errchk(hipFreeAsync(d_mask_zm, custream));
    cu_errchk(hipFreeAsync(d_mask_zm_fft, custream));
    cu_errchk(hipStreamSynchronize(custream));
    cu_errchk(hipStreamDestroy(custream));
}


void
oemd_coefficients_search::create_random_mask(
        const unsigned position, double step, const unsigned mask_size, std::vector<double> &mask, CPTR(double) start_mask,
        common::t_drand48_data_ptr buffer, hipfftHandle plan_mask_forward, hipfftHandle plan_mask_backward, const unsigned gpu_id)
{
    step *= common::drander(buffer);
    if (!start_mask) {
        UNROLL()
        for (unsigned i = 0; i < mask_size; ++i) mask[i] = common::drander(buffer);
    } else {
#pragma omp parallel for default(shared) num_threads(adj_threads(mask_size))
        for (unsigned i = 0; i < mask_size; ++i) {
            if (common::drander(buffer) > .25) {
                if (common::drander(buffer) > .05) {
                    if (common::drander(buffer) > .5) {
                        //masks[i]=start_mask[i]+step*(-log(drand48()));
                        mask[i] = start_mask[i] + step * common::drander(buffer);
                    } else {
                        mask[i] = std::max<double>(0., start_mask[i] - step * common::drander(buffer));
                    }
                } else {
                    mask[i] = start_mask[i] * (1. + step * (2 * common::drander(buffer) - 1));
                }
            } else {
                mask[i] = start_mask[i];
            }
        }
    }

    if (common::drander(buffer) > .01) gauss_smoothen_mask(mask_size, mask, buffer, plan_mask_forward, plan_mask_backward, gpu_id);

    // fix_mask(mask.data(), mask.data(), mask.size());
}


__global__ void compute_power_spectrum(CRPTR(hipfftDoubleComplex) d_freq, double *const d_psd, double *const d_psd_sum, const unsigned N, const unsigned N_2_1)
{
    CU_STRIDED_FOR_i(N_2_1) {
        d_psd[i] = fabs(d_freq[i].x) + fabs(d_freq[i].y);
        atomicAdd(d_psd_sum, d_psd[i]);
    }
}

__global__ void normalize_psd(double *const d_psd, CRPTR(double) d_psd_sum, const unsigned N_2_1)
{
    CU_STRIDED_FOR_i(N_2_1) d_psd[i] /= *d_psd_sum;
}

__global__ void compute_spectral_entropy(CRPTR(double) d_psd, double *const d_entropy, const unsigned N_2_1)
{
    CU_STRIDED_FOR_i(N_2_1) if (d_psd[i] > 0) atomicAdd(d_entropy, -d_psd[i] * log2f(d_psd[i]));
}

double oemd_coefficients_search::compute_spectral_entropy_cufft(double *d_signal, unsigned N, const hipStream_t custream)
{
    if (N > C_cufft_input_limit) {
        d_signal += N - C_cufft_input_limit;
        N = C_cufft_input_limit;
    } else if (N % 2) {
        ++d_signal;
        --N;
    }
    hipfftDoubleComplex *d_freq;
    double *d_psd;
    const auto N_2_1 = common::fft_len(N);

    // Allocate memory on the device
    cu_errchk(hipMallocAsync((void **) &d_freq, N_2_1 * sizeof(hipfftDoubleComplex), custream));
    cu_errchk(hipMallocAsync((void **) &d_psd, N_2_1 * sizeof(double), custream));
    auto d_psd_sum = cucalloc<double>(custream);
    auto d_entropy = cucalloc<double>(custream);

    // Create a CUFFT plan
    hipfftHandle plan;
    cf_errchk(hipfftPlan1d(&plan, N, HIPFFT_D2Z, 1));
    cf_errchk(hipfftSetStream(plan, custream));
    // Execute the FFT
    cf_errchk(hipfftExecD2Z(plan, (double *) d_signal, d_freq));

    // Compute the Power Spectral Density (PSD)
    compute_power_spectrum<<<CU_BLOCKS_THREADS(N_2_1), 0, custream>>>(d_freq, d_psd, d_psd_sum, N, N_2_1);

    // Normalize the PSD
    normalize_psd<<<CU_BLOCKS_THREADS(N_2_1), 0, custream>>>(d_psd, d_psd_sum, N_2_1);

    // Compute the Spectral Entropy
    compute_spectral_entropy<<<CU_BLOCKS_THREADS(N_2_1), 0, custream>>>(d_psd, d_entropy, N_2_1);

    // Copy the result back to the host
    double entropy;
    cu_errchk(hipMemcpyAsync((void **) &entropy, d_entropy, sizeof(double), hipMemcpyDeviceToHost, custream));

    // Clean up
    cu_errchk(hipFreeAsync((void *) d_freq, custream));
    cu_errchk(hipFreeAsync((void *) d_psd, custream));
    cu_errchk(hipFreeAsync((void *) d_entropy, custream));
    cu_errchk(hipStreamSynchronize(custream));
    hipfftDestroy(plan);

    return entropy;
}


void cu_normalize(double *const d_in, const unsigned n, const hipStream_t custream)
{
    const auto mean = solvers::mean(d_in, n, custream);
    if (mean != 0) oemd::G_subtract_inplace<<<CU_BLOCKS_THREADS(n), 0, custream>>>(d_in, mean, n);
    const auto meanabs = solvers::meanabs(d_in, n, custream);
    if (meanabs != 1) datamodel::G_div_inplace<<<CU_BLOCKS_THREADS(n), 0, custream>>>(d_in, meanabs, n);
}

template<typename T> __device__ inline T sinc(const T x)
{
    return x == T(0) ? 1 : sin(M_PI * x) / (M_PI * x);
}

__global__ void G_generate_fir_mask(RPTR(double) d_mask, const double f, const unsigned len, const double len_2, const double len_1, const double f_2)
{
    constexpr double alpha0 = .42;
    constexpr double alpha1 = .5;
    constexpr double alpha2 = .08;
    constexpr auto pi_2 = 2 * M_PI;
    constexpr auto pi_4 = 4 * M_PI;

    CU_STRIDED_FOR_i(len)d_mask[i] = /* low-pass */ f_2 * sinc(f_2 * (i - len_2)) * /* blackman */ (alpha0 - alpha1 * cos(pi_2 * i / len_1) - alpha2 * cos(pi_4 * i / len_1));
}

double *generate_fir_mask(const unsigned len, const double f, const hipStream_t custream)
{
    double *d_mask;
    cu_errchk(hipMallocAsync(&d_mask, len * sizeof(*d_mask), custream));
    G_generate_fir_mask<<<CU_BLOCKS_THREADS(len), 0, custream>>>(d_mask, f, len, len * .5, len - 1, 2 * f);
    return d_mask;
}

// TODO Port to CUDA
std::vector<double> lbp_fir(const double As_, const double fp, const double fs_, const double Fs)
{
    constexpr bool Kaiser = false;
    const auto fs = std::min<double>(Fs, fp + fs_);
    const auto As = _MIN(1, As_) * 74;

    // Cut-off frequency
    const double fc = (fs + fp) / 2;

    // Transition band (rad/sec)
    const double Tb = C_pi_2 * (fs - fp) / Fs;
    std::vector<double> w;
    unsigned N;

    // Choice of window function based on stopband attenuation
    if (Kaiser) {
        // Beta estimation
        double beta;
        if (As > 50)
            beta = 0.1102 * (As - 8.7);
        else if (21 < As && As <= 50)
            beta = 0.5842 * std::pow(As - 21, 0.4) + 0.07886 * (As - 21);
        else
            beta = 0;

        // Filter Order
        N = cdiv(As - 8, 2.285 * Tb);

        // Kaiser window function, w[n]
        w.resize(N);
        const auto I0_beta = boost::math::cyl_bessel_i(0., beta);
        const auto N_1 = N + 1;
        OMP_FOR_i(N) w[i] = boost::math::cyl_bessel_i(0., beta * std::sqrt(1. - std::pow((2. * i - N_1) / N, 2))) / I0_beta;
    } else {
        if (As <= 21) { // Rectangular
            N = cdiv(1.8 * M_PI, Tb);
            w = std::vector<double>(N, 1.0); // Rectangular window
        } else if (As > 21 && As <= 26) { // Bartlett
            N = cdiv(6.1 * M_PI, Tb);
            w.resize(N);
            OMP_FOR_i(N) w[i] = 1.0 - std::abs(2.0 * i / (N - 1) - 1.0); // Bartlett window
        } else if (As > 26 && As <= 44) { // Hann
            N = cdiv(6.2 * M_PI, Tb);
            w.resize(N);
            OMP_FOR_i(N) w[i] = 0.5 * (1 - std::cos(C_pi_2 * i / (N - 1))); // Hann window
        } else if (As > 44 && As <= 53) { // Hamming
            N = cdiv(6.6 * M_PI, Tb);
            w.resize(N);
            OMP_FOR_i(N) w[i] = 0.54 - 0.46 * std::cos(C_pi_2 * i / (N - 1)); // Hamming window
        } else if (As > 53) { // Blackman  // && As <= 74
            N = cdiv(11 * M_PI, Tb);
            w.resize(N);
            OMP_FOR_i(N) w[i] = 0.42 - 0.5 * std::cos(C_pi_2 * i / (N - 1)) + 0.08 * std::cos(4 * M_PI * i / (N - 1)); // Blackman window
        }
    }
    if (N > oemd_coefficients_search::C_fir_max_len) {
        LOG4_WARN("FIR mask too long " << N);
        return {};
    }

    // Ideal impulse response of lowpass filter
    const auto alpha = N / 2;
    const auto fc_2 = 2 * fc;
    const auto fc_2_pi = fc * C_pi_2;

    // Multiplying the ideal filter response to window function
    std::vector<double> FIR_lowpass(N);
    OMP_FOR_i(N) FIR_lowpass[i] = (i == alpha ? fc_2 / Fs : std::sin(fc_2_pi / Fs * (i - alpha)) / (M_PI * (i - alpha))) * w[i];

    LOG4_TRACE("Returning " << N << " FIR coefficients.");
    return FIR_lowpass;
}


__global__ void G_quantise_labels_quick(
        CRPTR(double) d_imf, RPTR(double) d_labels, const unsigned validate_len, const unsigned q_validate_len, const unsigned label_len,
        CRPTR(t_label_ix) d_label_ixs, CRPTR(unsigned) ix_end_F)
{
    CU_STRIDED_FOR_i(q_validate_len) {
        double v = 0;
        UNROLL(C_max_label_ixs / 100)
        for (auto j = 0; j < d_label_ixs[i].n_ixs; ++j) v += d_imf[d_label_ixs[i].label_ixs[j]];
        d_labels[i] = v / d_label_ixs[i].n_ixs;
#ifdef EMO_DIFF
        d_labels[i] -= d_imf[ix_end_F[i] - 1];
#endif
    }
}


double
oemd_coefficients_search::evaluate_mask(
        const double att, const double fp, const double fs, const std::vector<double> &workspace, const unsigned validate_start_ix,
        const unsigned validate_len, const unsigned siftings, const double meanabs_input,
        const std::vector<unsigned> &times, const std::vector<t_label_ix> &label_ixs, const std::deque<t_feat_params> &feat_params) const
{
    const auto mask = lbp_fir(att, fp, fs, sample_rate);
    if (mask.empty()) {
        LOG4_WARN("Bad mask for attenuation " << att << ", freq pass " << fp << ", freq stop " << fs);
        return common::C_bad_validation;
    }

    const auto full_input_len = workspace.size();
    const unsigned mask_len = mask.size();
    CTX_CUSTREAM;
    const auto d_mask = cumallocopy(mask, custream);
    const auto d_workspace = cumallocopy(workspace, custream);
    double *d_tmp;
    cu_errchk(hipMallocAsync((void **) &d_tmp, full_input_len * sizeof(double), custream));
    sift(siftings, full_input_len, mask_len, custream, d_mask, d_workspace, d_tmp);
    cu_errchk(hipFreeAsync(d_tmp, custream));
    cu_errchk(hipFreeAsync(d_mask, custream));

    const auto d_imf = d_workspace + validate_start_ix;
#if 1 // Component power
    const auto meanabs_imf = solvers::meanabs(d_imf, validate_len, custream);
    if (!std::isnormal(meanabs_imf)) {
        LOG4_WARN("Bad IMF " << meanabs_imf);
        cu_errchk(hipFreeAsync(d_workspace, custream));
        cu_sync_destroy(custream);
        return common::C_bad_validation;
    }
    const auto rel_pow = std::abs(meanabs_input / meanabs_imf - levels + 1.); // It was -1 for meanabs_input
#else
    constexpr double meanabs_imf = 1;
    constexpr double rel_pow = 1;
#endif

    double *d_labels, *d_features, *d_scores;
    const unsigned validate_rows = label_ixs.size();
    cu_errchk(hipMallocAsync(&d_labels, validate_rows * sizeof(*d_labels), custream));
    const auto d_label_ixs = cumallocopy(label_ixs, custream);
    std::vector<unsigned> ix_end_F(validate_rows);
    OMP_FOR_i(validate_rows) ix_end_F[i] = feat_params[i].ix_end;
    const auto d_ix_end_F = cumallocopy(ix_end_F, custream);
    G_quantise_labels_quick<<<CU_BLOCKS_THREADS(validate_rows), 0, custream>>>(d_imf, d_labels, validate_len, validate_rows, label_len, d_label_ixs, d_ix_end_F);
    cu_errchk(hipFreeAsync(d_label_ixs, custream));
    cu_errchk(hipFreeAsync(d_ix_end_F, custream));

    constexpr auto full_feat_cols = datamodel::C_features_superset_coef * datamodel::C_default_svrparam_lag_count;
    constexpr auto feat_cols_ileave = full_feat_cols / C_column_interleave;

    auto autocor = common::C_bad_validation;
    const unsigned cols_rows_q = validate_rows * feat_cols_ileave;
    const auto features_size = cols_rows_q * sizeof(*d_features);
    cu_errchk(hipMallocAsync((void **) &d_features, features_size, custream));
    cu_errchk(hipMallocAsync((void **) &d_scores, feat_cols_ileave * sizeof(*d_scores), custream));
    std::vector<t_feat_params> feat_params_q(feat_params.size());
    OMP_FOR_i(feat_params.size()) {
        feat_params_q[i].end_time = feat_params[i].end_time;
        feat_params_q[i].ix_end = feat_params[i].ix_end;
    }
    const auto d_times = cumallocopy(times.cbegin() + validate_start_ix, times.cend(), custream);

    UNROLL(business::ModelService::C_num_quantisations / C_quantisation_interleave)
    for (unsigned q = 0; q < business::ModelService::C_num_quantisations; q += C_quantisation_interleave) {
        const auto qt = business::ModelService::C_quantisations[q];
        OMP_FOR_i(feat_params.size()) {
            feat_params_q[i].time_start = feat_params[i].end_time - full_feat_cols * qt;
            feat_params_q[i].ix_start = before_bound(times.cbegin(), times.cend(), feat_params_q[i].time_start) - times.cbegin() - validate_start_ix;
        }
        const auto d_feat_params_q = cumallocopy(feat_params_q, custream);
        const auto interleave_qt = C_column_interleave * qt;
        cu_errchk(hipMemsetAsync(d_features, 0, features_size, custream));
        G_quantise_features<<<CU_BLOCKS_THREADS(validate_rows), 0, custream>>>(
                d_features, d_imf, d_times, d_feat_params_q, validate_rows, feat_cols_ileave, qt, interleave_qt, interleave_qt * .5);
        cu_errchk(hipFreeAsync(d_feat_params_q, custream));
        cu_align_features<<<CU_BLOCKS_THREADS(feat_cols_ileave), 0, custream>>>(d_features, d_labels, d_scores, nullptr, nullptr, nullptr, validate_rows, feat_cols_ileave);
        double score;
        if (feat_cols_ileave > datamodel::C_default_svrparam_lag_count) {
            thrust::sort(thrust::cuda::par.on(custream), d_scores, d_scores + feat_cols_ileave);
            score = solvers::sum(d_scores, datamodel::C_default_svrparam_lag_count, custream);
        } else
            score = solvers::sum(d_scores, feat_cols_ileave, custream);
        if (score < autocor) autocor = score;
    }
    cu_errchk(hipFreeAsync(d_times, custream));
    cu_errchk(hipFreeAsync(d_scores, custream));
    cu_errchk(hipFreeAsync(d_features, custream));
    cu_errchk(hipFreeAsync(d_labels, custream));

    // Spectral entropy
    const auto inv_entropy = 1.; // compute_spectral_entropy_cufft(d_imf, validate_len, custream);
    cu_errchk(hipFreeAsync(d_workspace, custream)); // d_imf is a part of d_workspace
    cu_sync_destroy(custream);

    // Weights and final score
    constexpr double autocor_w = 1;
    constexpr double rel_pow_w = 2;
    constexpr double inv_entropy_w = 1;
    const auto score = std::pow(rel_pow, rel_pow_w) * std::pow(autocor, autocor_w) * std::pow(inv_entropy, inv_entropy_w);
    LOG4_TRACE("Returning autocorrelation " << autocor << ", relative power " << rel_pow << ", score " << score << ", inv entropy " << inv_entropy << ", meanabs imf " <<
                                            meanabs_imf << ", meanabs input " << meanabs_input);
    return score;
}


void oemd_coefficients_search::sift(
        const unsigned siftings, const unsigned full_input_len, const unsigned mask_len, const hipStream_t custream, CPTR(double) d_mask, double *const d_rx,
        double *const d_rx2) const noexcept
{
    UNROLL()
    for (unsigned s = 0; s < siftings; ++s) {
        oemd::G_apply_fir<<<CU_BLOCKS_THREADS(full_input_len), 0, custream>>>(
                stretch_coef, d_rx, full_input_len, d_mask, mask_len, mask_len * stretch_coef, d_rx2, 0);
        oemd::G_subtract_inplace<<<CU_BLOCKS_THREADS(full_input_len), 0, custream>>>(d_rx, d_rx2, full_input_len);
    }
}

// Function to calculate the magnitude of complex numbers
__global__ void G_calculate_magnitude(CRPTR(hipfftDoubleComplex) freq_domain, RPTR(double) magnitudes, const unsigned N)
{
    CU_STRIDED_FOR_i(N) magnitudes[i] = fabs(freq_domain[i].x) + fabs(freq_domain[i].y);
}

// Find n-th percentile broadest and tallest peak in the vector
unsigned find_nth_peak(const std::vector<double> &data, const double n)
{
    // LOG4_DEBUG("Input magnitudes " << common::present(arma::vec(data)));
    std::map<double, size_t, common::safe_double_less> peaks;
    t_omp_lock peak_l;
    // Find all peaks in the vector
    OMP_FOR_i((unsigned) data.size()) {
        double peak_width = 0;
        // Check m neighbors
        for (unsigned j = 1; j < std::abs<int>(i - data.size()); ++j) {
            bool peak_left = false;
            bool peak_right = false;
            if (i < j)
                peak_left = true;
            else if (data[i] > data[i - j]) {
                peak_width += 1;
                peak_left = true;
            }
            if (i + j >= data.size())
                peak_right = true;
            else if (data[i] > data[i + j]) {
                peak_width += 1;
                peak_right = true;
            }
            if (!peak_right || !peak_left) break;
        }
        // If it is a peak, add to the list
        if (peak_width > 0) {
            peak_l.set();
            peaks.emplace(peak_width + data[i], i);
            peak_l.unset();
        }
    }
    if (peaks.empty()) LOG4_THROW("No peaks found.");
    const double size_1 = peaks.size() - 1;
    const auto res = *std::next(peaks.cbegin(), n * size_1);
    LOG4_DEBUG("Found " << peaks.size() << " peaks, starting " << *peaks.cbegin() << ", ending " << *peaks.rbegin() << ", returning " << n << " percentile, " << res);
    return res.second;
}

double oemd_coefficients_search::dominant_frequency(const std::vector<double> &input, const double percentile_greatest_peak, const hipStream_t custream) const
{
    const unsigned n = input.size() - input.size() % 2;
    const unsigned fft_n = common::fft_len(n);

    // Allocate device memory
    double *d_signal, *d_magnitudes;
    hipfftDoubleComplex *d_freq_domain;
    cu_errchk(hipMallocAsync(&d_signal, n * sizeof(*d_signal), custream));
    cu_errchk(hipMallocAsync(&d_magnitudes, fft_n * sizeof(*d_magnitudes), custream));
    cu_errchk(hipMallocAsync(&d_freq_domain, fft_n * sizeof(*d_freq_domain), custream));

    // Copy signal to device
    cu_errchk(hipMemcpyAsync(d_signal, input.data(), n * sizeof(*d_signal), hipMemcpyHostToDevice, custream));

    // Create CUFFT plan
    hipfftHandle plan;
    cf_errchk(hipfftPlan1d(&plan, n, HIPFFT_D2Z, 1));
    cf_errchk(hipfftSetStream(plan, custream));
    // Execute the plan
    cf_errchk(hipfftExecD2Z(plan, d_signal, d_freq_domain));

    // Calculate magnitudes of the frequency components
    G_calculate_magnitude<<<CU_BLOCKS_THREADS(fft_n), 0, custream>>>(d_freq_domain, d_magnitudes, fft_n);

    // Copy magnitudes back to host
    std::vector<double> magnitudes(fft_n);
    cu_errchk(hipMemcpyAsync(magnitudes.data(), d_magnitudes, fft_n * sizeof(*d_magnitudes), hipMemcpyDeviceToHost, custream));
    // Clean up
    cu_errchk(hipFreeAsync(d_signal, custream));
    cu_errchk(hipFreeAsync(d_freq_domain, custream));
    cu_errchk(hipFreeAsync(d_magnitudes, custream));
    cu_errchk(hipStreamSynchronize(custream));
    cf_errchk(hipfftDestroy(plan));
    return find_nth_peak(magnitudes, percentile_greatest_peak) * sample_rate / n;
}


void
oemd_coefficients_search::run(
        const datamodel::datarow_crange &input,
        const std::vector<double> &tail,
        std::deque<std::vector<double>> &masks,
        std::deque<unsigned> &siftings,
        const unsigned window_start,
        const unsigned window_end,
        const std::string &queue_name,
        const unsigned in_colix,
        const datamodel::t_iqscaler &scaler) const
{
    if (gpuids.empty()) LOG4_THROW("No GPUs found, aborting.");
    const auto gpu_id = gpuids.front();
    const auto window_len = window_end - window_start;
    const auto window_size = window_len * sizeof(double);
    assert(masks.size() == levels - 1);
#ifndef EMD_ONLY
    assert(in_colix == levels * 2);
#endif
    std::vector<double> workspace(window_len);
    std::deque<bpt::ptime> times(window_len);
    std::vector<unsigned> times_i(window_len);
    const auto first_time = input.front()->get_value_time();
    const auto first_time_t = bpt::to_time_t(first_time);
    OMP_FOR(window_len)
    for (unsigned i = window_start; i < window_end; ++i) {
        double value;
        bpt::ptime time;
        if (i < tail.size()) {
            value = tail[i];
            time = first_time - resolution * (tail.size() - i);
        } else {
            const auto p_row = input[i - tail.size()];
            value = p_row->at(in_colix);
            time = p_row->get_value_time();
        }
        workspace[i - window_start] = scaler(value);
        times[i - window_start] = time;
        times_i[i - window_start] = bpt::to_time_t(time) - first_time_t;
    }
    // LOG4_TRACE("Tail " << common::present(arma::vec(tail)) << ", workspace " << common::present(arma::vec(workspace)));

    const auto label_duration = label_len * resolution;
    const auto label_times = [&] {
        std::deque<bpt::ptime> r;
        UNROLL(16)
        for (boost::posix_time::ptime it_time(times.front().date(), bpt::hours(times.front().time_of_day().hours()) + onehour);
             it_time < times.back();
             it_time += label_duration)
            r.emplace_back(it_time);
        return r;
    }();
    const auto label_half_duration = label_duration * .5;
    const auto horizon_duration = label_duration * PROPS.get_prediction_horizon();
    std::deque<t_label_ix> label_ixs;
    std::deque<t_feat_params> feat_params;
    const unsigned horizon_samples_1 = label_len * PROPS.get_prediction_horizon() + 1;
    OMP_FOR_(label_times.size(), ordered)
    for (const auto &it_time: label_times) {
        const auto L_start_it = std::lower_bound(times.cbegin(), times.cend(), it_time);
        if (L_start_it == times.cend() || *L_start_it - it_time > label_half_duration) continue;
        const unsigned L_start_ix = L_start_it - times.cbegin();
        if (L_start_ix < max_row_len) continue;

        const auto L_end_time = it_time + label_duration;
        const auto L_end_it = std::lower_bound(L_start_it, std::min(L_start_it + label_len + 1, times.cend()), L_end_time);
        const unsigned L_end_ix = L_end_it - times.cbegin();

        const auto F_time = it_time - horizon_duration;
        auto F_end_it = lower_bound(std::max(L_start_it - horizon_samples_1, times.cbegin()), L_start_it, F_time);
        if (F_end_it == times.cend() || F_end_it == times.cbegin()) continue;
        const unsigned F_end_ix = F_end_it - times.cbegin();
        if (F_end_ix < max_row_len) continue;

        t_label_ix label_ix{label_len};
        const auto this_label_ixs = generate_twap_indexes(times.cbegin(), L_start_it, L_end_it, it_time, L_end_time, resolution, label_len);
        memcpy(label_ix.label_ixs, this_label_ixs.data(), label_len * sizeof(unsigned));
#pragma omp ordered
        {
            label_ixs.emplace_back(label_ix);
            feat_params.emplace_back(t_feat_params{.end_time = unsigned(bpt::to_time_t(F_time) - first_time_t), .ix_end = F_end_ix});
        };
    }
    assert(label_ixs.size() == feat_params.size());
    release_cont(times);

    LOG4_DEBUG(
            "Optimizing " << masks.size() << " masks for queue " << queue_name << " using " << gpuids.size() << " GPUs, tail len " << tail.size() << ", window len "
                          << window_len << ", window start " << window_start << ", window end " << window_end << ", levels " << levels << ", input column index " << in_colix <<
                          ", label ixs " << label_ixs.size() << ", first label last feature ix " << feat_params.front().ix_end);

    cu_errchk(hipSetDevice(gpu_id));
    hipStream_t custream;
    cu_errchk(hipStreamCreateWithFlags(&custream, C_cu_default_stream_flags));
    unsigned validation_start_ix = std::max<unsigned>(tail.size(), C_fir_max_len);
    UNROLL()
    for (unsigned m = 0; m < masks.size(); ++m) {
        const auto level = levels - m - 1;

        const unsigned in_window_len = workspace.size() - validation_start_ix;
        const auto meanabs_input = common::meanabs<double>(workspace.cbegin() + validation_start_ix, workspace.cend());

        const auto min_feat_ix = validation_start_ix + max_row_len;
        unsigned l_start;
        for (l_start = 0; l_start < feat_params.size(); ++l_start)
            if (feat_params[l_start].ix_end >= min_feat_ix) break;
        const auto feat_params_level_len = label_ixs.size() - l_start;
        std::vector<t_label_ix> label_ixs_level(feat_params_level_len);
        std::deque<t_feat_params> feat_params_level(feat_params_level_len);
        OMP_FOR_i(feat_params_level_len) {
            const auto i_l_start = i + l_start;
            t_label_ix label_ix = label_ixs[i_l_start];
            UNROLL(C_max_label_ixs / 100)
            for (auto &lix: label_ix.label_ixs) lix -= validation_start_ix;
            label_ixs_level[i] = label_ix;

            feat_params_level[i] = feat_params[i_l_start];
            feat_params_level[i].ix_end -= validation_start_ix;
        }

        LOG4_DEBUG(
                "Optimizing " << siftings[m] << " siftings, " << in_window_len << " window len, " << workspace.size() << " workspace len, valid start ix " << validation_start_ix <<
              ", level " << level << ", meanabs input " << meanabs_input << ", max quantisation " << business::ModelService::C_max_quantisation << ", level label ixs " <<
              label_ixs_level.size() << ", latest label last feature ix " << feat_params.back().ix_end << ", max row len " << max_row_len);

        const auto loss_function = [&, validation_start_ix, in_window_len, siftings, meanabs_input]
#ifdef USE_FIREFLY
                (const std::vector<double> &x) {
            return
#else
                (const double *x, double *const f) {
            *f =
#endif
                    evaluate_mask(x[0], x[1], x[2], workspace, validation_start_ix, in_window_len, siftings[m], meanabs_input, times_i, label_ixs_level, feat_params_level);
        };
        auto freq = common::constrain(dominant_frequency(workspace, .95, custream), 1. / in_window_len, 1.);
        arma::vec x0(3, arma::fill::none);
        x0[0] = .5;
        x0[1] = freq;
        x0[2] = .01;
        constexpr double freq_range = 1.25;
        arma::mat bounds(3, 2, arma::fill::none);
        bounds(0, 0) = 1e-1; // Min attenuation
        bounds(0, 1) = 1; // Max attenuation
        bounds(1, 0) = std::max(freq / freq_range, 1. / in_window_len); // Min pass frequency
        bounds(1, 1) = std::min(freq * freq_range, 1.); // Max pass frequency
        bounds(2, 0) = 5e-4; // Min frequency stop band
        bounds(2, 1) = 1; // Max frequency stop band
#ifdef USE_FIREFLY
        double score;
        std::tie(score, h_mask) = optimizer::firefly(
                h_mask.size(), particles, iterations, common::C_FFA_alpha, common::C_FFA_betamin, common::C_FFA_gamma, bounds,
                arma::vec(h_mask.size(), arma::fill::ones), loss_function).operator std::pair<double, std::vector<double>>();
        return score;
#else
        optimizer::pprune opt(0, particles, bounds, loss_function, 1, 0, 0, x0);
        const optimizer::t_pprune_res res = opt;
        masks[m] = lbp_fir(res.best_parameters[0], res.best_parameters[1], res.best_parameters[2], sample_rate);
        if (masks[m].empty()) LOG4_THROW("Bad mask for parameters " << res.best_parameters);
#endif
        const auto mask_len = masks[m].size();
        LOG4_DEBUG("Level " << level << ", mask " << m << ", queue " << queue_name << ", score " << res.best_score);
        save_mask(masks[m], queue_name, m, masks.size() + 1);
        cu_errchk(hipSetDevice(gpu_id));
        auto d_level_imf = cumallocopy(workspace, custream);
        const auto d_mask = cumallocopy(masks[m], custream);
        double *d_workspace;
        cu_errchk(hipMallocAsync((void **) &d_workspace, window_size, custream));
        transform(d_level_imf, d_mask, window_len, mask_len, siftings[m], d_workspace, custream);
        cu_errchk(hipFreeAsync(d_mask, custream));
        cu_errchk(hipMemcpyAsync(d_workspace, workspace.data(), window_size, hipMemcpyHostToDevice, custream));
        G_subtract_inplace<<<CU_BLOCKS_THREADS(window_len), 0, custream>>>(d_workspace, d_level_imf, window_len);
        cu_errchk(hipFreeAsync(d_level_imf, custream));
        validation_start_ix += siftings[m] * mask_len;
        cu_errchk(hipMemcpyAsync(workspace.data(), d_workspace, window_size, hipMemcpyDeviceToHost, custream));
        cu_errchk(hipFreeAsync(d_workspace, custream));
        cu_errchk(hipStreamSynchronize(custream));
    }
    cu_errchk(hipStreamDestroy(custream));
}

} // oemd_search
} // svr
